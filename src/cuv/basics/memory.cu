#include "memory.hpp"

#include <thrust/device_ptr.h>

#include <cuv/tools/cuv_general.hpp>
#include <cuv/tools/meta_programming.hpp>

namespace cuv {

namespace detail {

template<class value_type>
void copy(value_type* dst, const value_type* src, size_t size, host_memory_space, host_memory_space,
        hipStream_t stream) {
    memcpy(dst, src, size * sizeof(value_type));
}

template<class value_type, class value_type2>
void copy(value_type* dst, const value_type2* src, size_t size, host_memory_space, host_memory_space,
        hipStream_t stream) {
    for (size_t i = 0; i < size; i++)
        dst[i] = static_cast<value_type>(src[i]);
}

template<class value_type>
void copy(value_type* dst, const value_type* src, size_t size, host_memory_space, dev_memory_space,
        hipStream_t stream) {
    cuvSafeCall(hipMemcpyAsync(dst, src, size * sizeof(value_type), hipMemcpyDeviceToHost, stream));
    if (stream == 0) {
        cuvSafeCall(hipStreamSynchronize(stream));
    }
}
template<class value_type>
void copy(value_type* dst, const value_type* src, size_t size, dev_memory_space, dev_memory_space,
        hipStream_t stream) {
    cuvSafeCall(hipMemcpyAsync(dst, src, size * sizeof(value_type), hipMemcpyDeviceToDevice, stream));
    if (stream == 0) {
        cuvSafeCall(hipStreamSynchronize(stream));
    }
}

template<class value_type, class value_type2>
void copy(value_type* dst, const value_type2* src, size_t size, host_memory_space, dev_memory_space,
        hipStream_t stream) {
    cuvSafeCall(hipMemcpyAsync(dst, src, size * sizeof(value_type), hipMemcpyDeviceToHost, stream));
    if (stream == 0) {
        cuvSafeCall(hipStreamSynchronize(stream));
    }
}

template<class value_type>
void copy(value_type* dst, const value_type* src, size_t size, dev_memory_space, host_memory_space,
        hipStream_t stream) {
    cuvSafeCall(hipMemcpyAsync(dst, src, size * sizeof(value_type), hipMemcpyHostToDevice, stream));
    if (stream == 0) {
        cuvSafeCall(hipStreamSynchronize(stream));
    }
}

template<class value_type, class value_type2>
void copy(value_type* dst, const value_type2* src, size_t size, dev_memory_space, dev_memory_space,
        hipStream_t stream) {
    if (IsSame<value_type, value_type2>::Result::value) {
        cuvSafeCall(hipMemcpyAsync(dst, src, size * sizeof(value_type), hipMemcpyDeviceToDevice, stream));
        if (stream == 0) {
            cuvSafeCall(hipStreamSynchronize(stream));
        }
    } else {
        thrust::copy(thrust::device_ptr<value_type2>(const_cast<value_type2*>(src)),
                thrust::device_ptr<value_type2>(const_cast<value_type2*>(src)) + size,
                thrust::device_ptr<value_type>(dst));
        cuvSafeCall(hipDeviceSynchronize());
    }
}

template<class value_type, class value_type2>
void copy2d(value_type* dst, const value_type2* src, size_t dpitch, size_t spitch, size_t h, size_t w,
        host_memory_space, host_memory_space, hipStream_t stream) {
    cuvSafeCall(hipMemcpy2DAsync(dst, dpitch * sizeof(value_type),
            src, spitch * sizeof(value_type2),
            w * sizeof(value_type), h, hipMemcpyHostToHost, stream));
    if (stream == 0) {
        cuvSafeCall(hipStreamSynchronize(stream));
    }
}

template<class value_type, class value_type2>
void copy2d(value_type* dst, const value_type2* src, size_t dpitch, size_t spitch, size_t h,
        size_t w, host_memory_space, dev_memory_space, hipStream_t stream) {
    cuvSafeCall(hipMemcpy2DAsync(dst, dpitch * sizeof(value_type), src, spitch * sizeof(value_type2),
            w * sizeof(value_type), h, hipMemcpyDeviceToHost, stream));
    if (stream == 0) {
        cuvSafeCall(hipStreamSynchronize(stream));
    }
}

template<class value_type, class value_type2>
void copy2d(value_type* dst, const value_type2* src, size_t dpitch, size_t spitch, size_t h,
        size_t w, dev_memory_space, host_memory_space, hipStream_t stream) {
    cuvSafeCall(hipMemcpy2DAsync(dst, dpitch * sizeof(value_type), src, spitch * sizeof(value_type2),
            w * sizeof(value_type), h, hipMemcpyHostToDevice, stream));
    if (stream == 0) {
        cuvSafeCall(hipStreamSynchronize(stream));
    }
}

template<class value_type, class value_type2>
void copy2d(value_type* dst, const value_type2* src, size_t dpitch, size_t spitch, size_t h,
        size_t w, dev_memory_space, dev_memory_space, hipStream_t stream) {
    cuvSafeCall(hipMemcpy2DAsync(dst, dpitch * sizeof(value_type),
            src, spitch * sizeof(value_type2),
            w * sizeof(value_type), h, hipMemcpyDeviceToDevice, stream));
    if (stream == 0) {
        cuvSafeCall(hipStreamSynchronize(stream));
    }
}

#define CUV_MEMORY_COPY(TYPE) \
template void copy<TYPE>(TYPE*, const TYPE*, size_t, host_memory_space, host_memory_space, hipStream_t); \
template void copy<TYPE>(TYPE*, const TYPE*, size_t, host_memory_space, dev_memory_space, hipStream_t); \
template void copy<TYPE>(TYPE*, const TYPE*, size_t, dev_memory_space, host_memory_space, hipStream_t); \
template void copy<TYPE>(TYPE*, const TYPE*, size_t, dev_memory_space, dev_memory_space, hipStream_t); \
template void copy2d<TYPE, TYPE>(TYPE*, const TYPE*, size_t, size_t, size_t, size_t, host_memory_space, host_memory_space, hipStream_t); \
template void copy2d<TYPE, TYPE>(TYPE*, const TYPE*, size_t, size_t, size_t, size_t, host_memory_space, dev_memory_space, hipStream_t); \
template void copy2d<TYPE, TYPE>(TYPE*, const TYPE*, size_t, size_t, size_t, size_t, dev_memory_space, host_memory_space, hipStream_t); \
template void copy2d<TYPE, TYPE>(TYPE*, const TYPE*, size_t, size_t, size_t, size_t, dev_memory_space, dev_memory_space, hipStream_t);

CUV_MEMORY_COPY(signed char);
CUV_MEMORY_COPY(unsigned char);
CUV_MEMORY_COPY(short);
CUV_MEMORY_COPY(unsigned short);
CUV_MEMORY_COPY(int);
CUV_MEMORY_COPY(unsigned int);
CUV_MEMORY_COPY(float);
CUV_MEMORY_COPY(double);

}

}
