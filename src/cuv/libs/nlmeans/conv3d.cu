#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and
 * proprietary rights in and to this software and related documentation and
 * any modifications thereto.  Any use, reproduction, disclosure, or distribution
 * of this software and related documentation without an express license
 * agreement from NVIDIA Corporation is strictly prohibited.
 *
 */


#include <cuv/basics/tensor.hpp>
#include <cuv/tools/cuv_general.hpp>
/*#include <math_functions.h>*/
// #include <vector>

namespace cuv{
	namespace libs{
		namespace nlmeans{
			inline unsigned int __host__ __device__ divup(unsigned int a, unsigned int b)
			{
				if (a % b)  /* does a divide b leaving a remainder? */
					return a / b + 1; /* add in additional block */
				else
					return a / b; /* divides cleanly */
			}


#define PITCH(PTR,P,Y,X) ((typeof(PTR))(PTR + ((size_t)P)*((size_t)Y)) + ((size_t)X))
#define MAX_KERNEL_RADIUS 8
#define      MAX_KERNEL_W (2 * MAX_KERNEL_RADIUS + 1)
			////////////////////////////////////////////////////////////////////////////////
			// Constants
			////////////////////////////////////////////////////////////////////////////////
#if 1
#  define   ROWS_BLOCKDIM_X 16
#  define   ROWS_BLOCKDIM_Y 16
#  define   ROWS_RESULT_STEPS 8
#  define   ROWS_HALO_STEPS 3
#else
#  define   ROWS_BLOCKDIM_X 16
#  define   ROWS_BLOCKDIM_Y 4
#  define   ROWS_RESULT_STEPS 8
#  define   ROWS_HALO_STEPS 1
#endif

#if 0
// original SDK
#  define   COLUMNS_BLOCKDIM_X 16
#  define   COLUMNS_BLOCKDIM_Y 8
#  define   COLUMNS_RESULT_STEPS 8
#  define   COLUMNS_HALO_STEPS 1
#else
#  define   COLUMNS_BLOCKDIM_X 16
#  define   COLUMNS_BLOCKDIM_Y 16
#  define   COLUMNS_RESULT_STEPS 8
#  define   COLUMNS_HALO_STEPS 3
#endif

#define   DEPTH_BLOCKDIM_Y 16
#define   DEPTH_BLOCKDIM_X 16
#define   DEPTH_RESULT_STEPS 4
#define   DEPTH_HALO_STEPS 3
			////////////////////////////////////////////////////////////////////////////////
			// Convolution kernel storage
			////////////////////////////////////////////////////////////////////////////////
			__constant__ float c_Kernel_h[MAX_KERNEL_W];

			void setConvolutionKernel(const cuv::tensor<float,dev_memory_space>&src){
				cuvSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_Kernel_h), const_cast<float*>(src.ptr()), src.size() * sizeof(float), 0,  hipMemcpyDeviceToDevice));
			}




			////////////////////////////////////////////////////////////////////////////////
			// Row convolution filter
			////////////////////////////////////////////////////////////////////////////////
		template<class SrcT, class DstT>
		__global__ void convolutionRowGPU(
				DstT *d_Dst,
				const SrcT *d_Src,
				int imageW,
				int imageH,
				int dpitch,
				int spitch,
				int KERNEL_RADIUS
				){
			__shared__ SrcT s_Data[ROWS_BLOCKDIM_Y][(ROWS_RESULT_STEPS + 2 * ROWS_HALO_STEPS) * ROWS_BLOCKDIM_X];

			//Offset to the left halo edge
			const int baseX = (blockIdx.x * ROWS_RESULT_STEPS - ROWS_HALO_STEPS) * ROWS_BLOCKDIM_X + threadIdx.x;
			const int baseY = blockIdx.y * ROWS_BLOCKDIM_Y + threadIdx.y;
			if(baseY>=imageH)
				return;

			d_Src = PITCH(d_Src, spitch, baseY, baseX);
			d_Dst = PITCH(d_Dst, dpitch, baseY, baseX);

			//Load main data
#pragma unroll
		       for(int i = ROWS_HALO_STEPS; i < ROWS_HALO_STEPS + ROWS_RESULT_STEPS; i++)
			       s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X] = (imageW - baseX > i * ROWS_BLOCKDIM_X)  ? d_Src[i * ROWS_BLOCKDIM_X] : 0;

			//Load left halo
#pragma unroll
		       for(int i = 0; i < ROWS_HALO_STEPS; i++)
			       s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X] = (baseX >= -i * ROWS_BLOCKDIM_X ) ? d_Src[i * ROWS_BLOCKDIM_X] : 0;

			//Load right halo
#pragma unroll
		       for(int i = ROWS_HALO_STEPS + ROWS_RESULT_STEPS; i < ROWS_HALO_STEPS + ROWS_RESULT_STEPS + ROWS_HALO_STEPS; i++)
			       s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X] = (imageW - baseX > i * ROWS_BLOCKDIM_X) ? d_Src[i * ROWS_BLOCKDIM_X] : 0;

			//Compute and store results
			__syncthreads();

/*#pragma unroll*/
			for(int i = ROWS_HALO_STEPS; i < ROWS_HALO_STEPS + ROWS_RESULT_STEPS; i++){
				float sum = 0;

/*#pragma unroll*/
				for(int j = -KERNEL_RADIUS; j <= KERNEL_RADIUS; j++){
					sum += s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X + j] * c_Kernel_h[KERNEL_RADIUS - j];
				}

				if(imageW - baseX > i * ROWS_BLOCKDIM_X)
					d_Dst[i * ROWS_BLOCKDIM_X] = sum;
			}
		}
			void convolutionRows(
					cuv::tensor<float,dev_memory_space> &dst,
					const cuv::tensor<float,dev_memory_space> &src,
					const cuv::tensor<float,dev_memory_space> &kernel
					){

                unsigned int kernel_radius = (kernel.size()-1) / 2;
                setConvolutionKernel(kernel);

				cuvAssert(equal_shape(dst,src));
				cuvAssert(kernel_radius <= MAX_KERNEL_RADIUS);
				int dw = src.shape()[2];
				int dh = src.shape()[1];
				int dd = src.shape()[0];
				dim3 blocks(divup(dw , (ROWS_RESULT_STEPS * ROWS_BLOCKDIM_X)), divup(dh*dd , ROWS_BLOCKDIM_Y));
				dim3 threads(ROWS_BLOCKDIM_X, ROWS_BLOCKDIM_Y);
				convolutionRowGPU<<<blocks, threads>>>(  dst.ptr(),  src.ptr(), dw, dh*dd,dst.stride(1),src.stride(1),kernel_radius);
				cuvSafeCall(hipDeviceSynchronize());
				safeThreadSync();
			}



			////////////////////////////////////////////////////////////////////////////////
			// Column convolution filter
			////////////////////////////////////////////////////////////////////////////////
			__global__ void convolutionColumnsKernel(
					float *d_Dst,
					const float *d_Src,
					int imageW,
					int imageH,
					int imageD,
					int pitch,
					int kernel_radius
					){
				__shared__ float s_Data[COLUMNS_BLOCKDIM_X][(COLUMNS_RESULT_STEPS + 2 * COLUMNS_HALO_STEPS) * COLUMNS_BLOCKDIM_Y + 1];

				int n_blocks_per_column = divup(imageH,(COLUMNS_RESULT_STEPS * COLUMNS_BLOCKDIM_Y));
				int baseZ = floor(float(blockIdx.y)/n_blocks_per_column);
				int blocky = blockIdx.y - baseZ*n_blocks_per_column;

				//Offset to the upper halo edge
				const int baseX = blockIdx.x * COLUMNS_BLOCKDIM_X + threadIdx.x;
				const int baseY = (blocky * COLUMNS_RESULT_STEPS - COLUMNS_HALO_STEPS) * COLUMNS_BLOCKDIM_Y + threadIdx.y;
				if(baseZ>=imageD) return;
				if(baseX>=imageW) return;
				if(baseY>=imageH) return;
				d_Src += baseZ*imageH*imageW + baseY * imageH + baseX;
				d_Dst += baseZ*imageH*imageW + baseY * imageH + baseX;

				//Main data
#pragma unroll
				for(int i = COLUMNS_HALO_STEPS; i < COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS; i++)
					s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y] =(imageH-baseY > i * COLUMNS_BLOCKDIM_Y) ? d_Src[i * COLUMNS_BLOCKDIM_Y * pitch] : 0;

				//Upper halo
				for(int i = 0; i < COLUMNS_HALO_STEPS; i++)
					s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y] =
						(baseY >= -i * COLUMNS_BLOCKDIM_Y) ? d_Src[i * COLUMNS_BLOCKDIM_Y * pitch] : 0;

				//Lower halo
				for(int i = COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS; i < COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS + COLUMNS_HALO_STEPS; i++)
					s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y] =
						(imageH - baseY > i * COLUMNS_BLOCKDIM_Y) ? d_Src[i * COLUMNS_BLOCKDIM_Y * pitch] : 0;

				//Compute and store results
				__syncthreads();
				// #pragma unroll
				for(int i = COLUMNS_HALO_STEPS; i < COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS; i++){
					float sum = 0;
#pragma unroll
					for(int j = -kernel_radius; j <= kernel_radius; j++)
						sum += c_Kernel_h[kernel_radius - j] * s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y + j];

					if(imageH-baseY > i * COLUMNS_BLOCKDIM_Y)
						d_Dst[i * COLUMNS_BLOCKDIM_Y * pitch] = sum;
				}
			}

			void convolutionColumns(
					cuv::tensor<float,dev_memory_space> & d_Dst,
					const cuv::tensor<float,dev_memory_space> & d_Src,
					const cuv::tensor<float,dev_memory_space> &kernel
					){

                unsigned int kernel_radius = (kernel.size()-1) / 2;
                setConvolutionKernel(kernel);

				int imageW = d_Src.shape()[2];
				int imageH = d_Src.shape()[1];
				int imageD = d_Src.shape()[0];
				cuvAssert( COLUMNS_BLOCKDIM_Y * COLUMNS_HALO_STEPS >= kernel_radius );
				/*cuvAssert( imageW % COLUMNS_BLOCKDIM_X == 0 );*/
				/*cuvAssert( imageH % (COLUMNS_RESULT_STEPS * COLUMNS_BLOCKDIM_Y) == 0 );*/

				dim3 blocks(divup(imageW , COLUMNS_BLOCKDIM_X), imageD * divup(imageH , COLUMNS_RESULT_STEPS * COLUMNS_BLOCKDIM_Y));
				dim3 threads(COLUMNS_BLOCKDIM_X, COLUMNS_BLOCKDIM_Y);

				convolutionColumnsKernel<<<blocks, threads>>>(
						d_Dst.ptr(),
						d_Src.ptr(),
						imageW,
						imageH,
						imageD,
						imageW,
						kernel_radius
						);
				cuvSafeCall(hipDeviceSynchronize());
			}

		template<class SrcT, class DstT>
		__global__ void convolutionDepthGPU(
				DstT *d_Dst,
				const SrcT *d_Src,
				int imageW,
				int imageH,
				size_t dpitch,
				size_t spitch,
				int KERNEL_RADIUS
				){
			__shared__ float s_Data[COLUMNS_BLOCKDIM_X][(COLUMNS_RESULT_STEPS + 2 * COLUMNS_HALO_STEPS) * COLUMNS_BLOCKDIM_Y + 1];

			//Offset to the upper halo edge
			const int baseX =  blockIdx.x * COLUMNS_BLOCKDIM_X + threadIdx.x;
			const int baseY = (blockIdx.y * COLUMNS_RESULT_STEPS - COLUMNS_HALO_STEPS) * COLUMNS_BLOCKDIM_Y + threadIdx.y;
			if(baseX>=imageW)
			       return;
			d_Src = PITCH(d_Src, spitch, baseY, baseX);
			d_Dst = PITCH(d_Dst, dpitch, baseY, baseX);

			//Main data
#pragma unroll
			for(int i = COLUMNS_HALO_STEPS; i < COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS; i++)
				s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y] = (imageH-baseY > i * COLUMNS_BLOCKDIM_Y) ? *PITCH(d_Src, spitch, i*COLUMNS_BLOCKDIM_Y,0) : 0;

			//Upper halo
#pragma unroll
			for(int i = 0; i < COLUMNS_HALO_STEPS; i++)
				s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y] = (baseY >= -i * COLUMNS_BLOCKDIM_Y) ? *PITCH(d_Src,spitch,i*COLUMNS_BLOCKDIM_Y,0) : 0;

			//Lower halo
#pragma unroll
			for(int i = COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS; i < COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS + COLUMNS_HALO_STEPS; i++)
				s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y]= (imageH - baseY > i * COLUMNS_BLOCKDIM_Y) ? *PITCH(d_Src,spitch,i*COLUMNS_BLOCKDIM_Y,0) : 0;

			//Compute and store results
			__syncthreads();
/*#pragma unroll*/
			for(int i = COLUMNS_HALO_STEPS; i < COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS; i++){
				float sum = 0;
/*#pragma unroll*/
				for(int j = -KERNEL_RADIUS; j <= KERNEL_RADIUS; j++)
					sum += s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y + j] * c_Kernel_h[KERNEL_RADIUS - j];
				if(imageH - baseY > i * COLUMNS_BLOCKDIM_Y)
					*PITCH(d_Dst,dpitch,i*COLUMNS_BLOCKDIM_Y,0) = sum;
					/**PITCH(d_Dst,dpitch,i*COLUMNS_BLOCKDIM_Y,0) = *PITCH(d_Src,spitch,i*COLUMNS_BLOCKDIM_Y,0);*/
					/**PITCH(d_Dst,dpitch,i*COLUMNS_BLOCKDIM_Y,0) = baseY+i*COLUMNS_BLOCKDIM_Y;*/
			}
		}


			void convolutionDepth(
					cuv::tensor<float,dev_memory_space>& dst,
					const cuv::tensor<float,dev_memory_space>& src,
					const cuv::tensor<float,dev_memory_space> &kernel
					){
                unsigned int kernel_radius = (kernel.size()-1) / 2;
                setConvolutionKernel(kernel);

				int dw = src.shape(2)*src.shape(1);
				int dh = src.shape(0);
				size_t spitch = src.stride(1)*src.shape(1);
				size_t dpitch = dst.stride(1)*dst.shape(1);

				cuvAssert( COLUMNS_BLOCKDIM_X * COLUMNS_HALO_STEPS >= kernel_radius );

				dim3 blocks(divup(dw , COLUMNS_BLOCKDIM_X), divup(dh , (COLUMNS_RESULT_STEPS * COLUMNS_BLOCKDIM_Y)));
				dim3 threads(COLUMNS_BLOCKDIM_X, COLUMNS_BLOCKDIM_Y);
				convolutionDepthGPU<<<blocks, threads>>>(  dst.ptr(),  src.ptr(), dw, dh, dpitch, spitch, kernel_radius);

				cuvSafeCall(hipDeviceSynchronize());
			}



			////////////////////////////////////////////////////////////////////////////////
			// Computes the higher eigenvalue of the hessian
			////////////////////////////////////////////////////////////////////////////////
			__device__ float computeDeterminant
				(float e00, float e01, float e02,
				 float e10, float e11, float e12,
				 float e20, float e21, float e22)
				{
					return e00*e11*e22-e00*e12*e21+e10*e21*e02-e10*e01*e22+e20*e01*e12-e20*e11*e02;
				}


			__global__ void hessianKernel
				(
				 float *d_output,
				 const float *d_gxx,
				 const float *d_gxy,
				 const float *d_gxz,
				 const float *d_gyy,
				 const float *d_gyz,
				 const float *d_gzz,
				 float sigma,
				 int imageW,
				 int imageH,
				 int imageD
				)
				{
					int n_blocks_per_width = imageW/blockDim.x;
					int z = (int)ceilf(blockIdx.x/n_blocks_per_width);
					int y = blockIdx.y*blockDim.y + threadIdx.y;
					int x = (blockIdx.x - z*n_blocks_per_width)*blockDim.x + threadIdx.x;
					int i = z*imageW*imageH + y*imageW + x;

					// // //Brute force eigen-values computation
					float a0, b0, c0, e0, f0, k0;
					a0 = -d_gxx[i]; b0 = -d_gxy[i]; c0 = -d_gxz[i];
					e0 = -d_gyy[i]; f0 = -d_gyz[i]; k0 = -d_gzz[i];


					// http://en.wikipedia.org/wiki/Eigenvalue_algorithm
					//Oliver K. Smith: Eigenvalues of a symmetric 3 × 3 matrix. Commun. ACM 4(4): 168 (1961)
					float m = (a0+e0+k0)/3;
					float q = computeDeterminant
						(a0-m, b0, c0, b0, e0-m, f0, c0, f0, k0-m)/2;
					float p = (a0-m)*(a0-m) + b0*b0 + c0*c0 + b0*b0 + (e0-m)*(e0-m) +
						f0*f0 + c0*c0 + f0*f0 + (k0-m)*(k0-m);
					p = p / 6;
					float phi = 1.f/3.f*atan(sqrt(p*p*p-q*q)/q);
					if(phi<0)
						phi=phi+3.14159f/3;

					float eig1 = m + 2*sqrt(p)*cos(phi);
					float eig2 = m - sqrt(p)*(cos(phi) + sqrt(3.0f)*sin(phi));
					float eig3 = m - sqrt(p)*(cos(phi) - sqrt(3.0f)*sin(phi));

					if( (eig1 > eig2) & (eig1 > eig3))
						d_output[i] = eig1*sigma*sigma;
					if( (eig2 > eig1) & (eig2 > eig3))
						d_output[i] = eig2*sigma*sigma;
					if( (eig3 > eig2) & (eig3 > eig1))
						d_output[i] = eig3*sigma*sigma;
				}



			void hessian
				(
				 cuv::tensor<float,dev_memory_space>& d_output,
				 const cuv::tensor<float,dev_memory_space>& d_gxx,
				 const cuv::tensor<float,dev_memory_space>& d_gxy,
				 const cuv::tensor<float,dev_memory_space>& d_gxz,
				 const cuv::tensor<float,dev_memory_space>& d_gyy,
				 const cuv::tensor<float,dev_memory_space>& d_gyz,
				 const cuv::tensor<float,dev_memory_space>& d_gzz,
				 float sigma
				)
				{
					int imageW = d_gxx.shape()[2];
					int imageH = d_gxx.shape()[1];
					int imageD = d_gxx.shape()[0];
					dim3 grid (imageD*imageW/ROWS_BLOCKDIM_X,imageH/ROWS_BLOCKDIM_Y);
					dim3 block(ROWS_BLOCKDIM_X,ROWS_BLOCKDIM_Y);
					hessianKernel<<<grid, block>>>( d_output.ptr(), d_gxx.ptr(), d_gxy.ptr(), d_gxz.ptr(),
							d_gyy.ptr(), d_gyz.ptr(), d_gzz.ptr(), sigma, imageW, imageH, imageD );
					cuvSafeCall(hipDeviceSynchronize());
				}



			/*********************************************************************************
			 ** hessian con orientacion
			 ********************************************************************************/
			__global__ void hessianKernelO
				(
				 float *d_output,
				 float *d_output_theta,
				 float *d_output_phi,
				 const float *d_gxx,
				 const float *d_gxy,
				 const float *d_gxz,
				 const float *d_gyy,
				 const float *d_gyz,
				 const float *d_gzz,
				 float sigma,
				 int imageW,
				 int imageH,
				 int imageD
				)
				{
					int n_blocks_per_width = imageW/blockDim.x;
					int z = (int)ceilf(blockIdx.x/n_blocks_per_width);
					int y = blockIdx.y*blockDim.y + threadIdx.y;
					int x = (blockIdx.x - z*n_blocks_per_width)*blockDim.x + threadIdx.x;
					int i = z*imageW*imageH + y*imageW + x;

					// // //Brute force eigen-values computation
					// http://en.wikipedia.org/wiki/Eigenvalue_algorithm
					//Oliver K. Smith: Eigenvalues of a symmetric 3 × 3 matrix. Commun. ACM 4(4): 168 (1961)
					float a0, b0, c0, d0, e0, f0;
					a0 = -d_gxx[i]; b0 = -d_gxy[i]; c0 = -d_gxz[i];
					d0 = -d_gyy[i]; e0 = -d_gyz[i]; f0 = -d_gzz[i];

					float m = (a0+d0+f0)/3;
					float q = computeDeterminant
						(a0-m, b0, c0, b0, d0-m, e0, c0, e0, f0-m)/2;
					float p = (a0-m)*(a0-m) + b0*b0 + c0*c0 + b0*b0 + (d0-m)*(d0-m) +
						e0*e0 + c0*c0 + e0*e0 + (f0-m)*(f0-m);
					p = p / 6;
					float phi = 1.f/3.f*atan(sqrt(p*p*p-q*q)/q);
					if(phi<0)
						phi=phi+3.14159f/3;

					float eig1 = m + 2*sqrt(p)*cos(phi);
					float eig2 = m - sqrt(p)*(cos(phi) + sqrt(3.0f)*sin(phi));
					float eig3 = m - sqrt(p)*(cos(phi) - sqrt(3.0f)*sin(phi));

					if( (eig1 > eig2) & (eig1 > eig3))
						d_output[i] = eig1*sigma*sigma;
					if( (eig2 > eig1) & (eig2 > eig3))
						d_output[i] = eig2*sigma*sigma;
					if( (eig3 > eig2) & (eig3 > eig1))
						d_output[i] = eig3*sigma*sigma;


					// // Now it comes to compute the eigenvector
					float l = d_output[i]/(sigma*sigma);
					a0 = a0 - l;
					d0 = d0 - l;
					f0 = f0 - l;
					float xv = b0*e0 - c0*d0;
					float yv = e0*a0 - c0*b0;
					float zv = d0*a0 - b0*b0;
					float radius = sqrt(xv*xv+yv*yv+zv*zv);
					float thetav = atan2(yv, xv);
					float phiv = 0;
					if(radius > 1e-6f)
						phiv = acos( zv/radius);

					d_output_theta[i] = thetav;
					d_output_phi[i] = phiv;

				}


			void hessian_orientation
				(
				 cuv::tensor<float,dev_memory_space> &d_Output,
				 cuv::tensor<float,dev_memory_space> &d_Output_theta,
				 cuv::tensor<float,dev_memory_space> &d_Output_phi,
				 const cuv::tensor<float,dev_memory_space> &d_gxx,
				 const cuv::tensor<float,dev_memory_space> &d_gxy,
				 const cuv::tensor<float,dev_memory_space> &d_gxz,
				 const cuv::tensor<float,dev_memory_space> &d_gyy,
				 const cuv::tensor<float,dev_memory_space> &d_gyz,
				 const cuv::tensor<float,dev_memory_space> &d_gzz,
				 float sigma
				)
				{
					int imageW = d_gxx.shape()[2];
					int imageH = d_gxx.shape()[1];
					int imageD = d_gxx.shape()[0];
					dim3 grid (imageD*imageW/ROWS_BLOCKDIM_X,imageH/ROWS_BLOCKDIM_Y);
					dim3 block(ROWS_BLOCKDIM_X,ROWS_BLOCKDIM_Y);
					hessianKernelO<<<grid, block>>>( d_Output.ptr(), d_Output_theta.ptr(), d_Output_phi.ptr(),
							d_gxx.ptr(), d_gxy.ptr(), d_gxz.ptr(),
							d_gyy.ptr(), d_gyz.ptr(), d_gzz.ptr(),
							sigma, imageW, imageH, imageD );
					cuvSafeCall(hipDeviceSynchronize());
				}

		}
	}
}
