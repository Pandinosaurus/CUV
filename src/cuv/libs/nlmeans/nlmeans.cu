#include "hip/hip_runtime.h"
//*LB*
// Copyright (c) 2010, University of Bonn, Institute for Computer Science VI
// All rights reserved.
// 
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
// 
//  * Redistributions of source code must retain the above copyright notice,
//    this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright notice,
//    this list of conditions and the following disclaimer in the documentation
//    and/or other materials provided with the distribution.
//  * Neither the name of the University of Bonn 
//    nor the names of its contributors may be used to endorse or promote
//    products derived from this software without specific prior written
//    permission.
// 
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
// ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
// WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
// FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
// DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
// SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
// OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//*LE*

#include <cstdio>
#include <cuv/tools/cuv_general.hpp>
#include <cuv/tools/progressbar.hpp>
#include <cuv/basics/cuda_array.hpp>
#include <cuv/basics/tensor.hpp>
#include <cuv/tensor_ops/tensor_ops.hpp>

#include "nlmeans.hpp"
#include "conv3d.hpp"

namespace cuv{
namespace libs{
namespace nlmeans{
#define PITCH(PTR,PITCH,Y,X) ((typeof(PTR))((PTR) + (PITCH)*(Y)) + (X))

	texture<float,         2, hipReadModeElementType> cuda_array_tex_float2d; 
	texture<unsigned char, 2, hipReadModeElementType> cuda_array_tex_uchar2d; 
	texture<float,         3, hipReadModeElementType> cuda_array_tex_float3d; 
	texture<unsigned char, 3, hipReadModeElementType> cuda_array_tex_uchar3d; 

	template<int dim, class T> struct texref{ };
	template<> struct texref<2,float>{
		typedef texture<float, 2, hipReadModeElementType> type;
		static 	inline __device__ __host__ type get(){ return cuda_array_tex_float2d; }; 
	};
	template<> struct texref<2,unsigned char>{
		typedef texture<unsigned char, 2, hipReadModeElementType> type;
		static inline __device__ __host__ type get(){ return cuda_array_tex_uchar2d; }; 
	};
	template<> struct texref<3,float>{
		typedef texture<float, 3, hipReadModeElementType> type;
		static 	inline __device__ __host__ type get(){ return cuda_array_tex_float3d; }; 
	};
	template<> struct texref<3,unsigned char>{
		typedef texture<unsigned char, 3, hipReadModeElementType> type;
		static inline __device__ __host__ type get(){ return cuda_array_tex_uchar3d; }; 
	};

	template<class V, class I1>
	__device__ I1 clamp(const V& i, const I1&maxi){
		return ((i<0)?0:((i>=maxi)?(maxi-1):i));
	}
	template<bool weights2d, class DstT, class SrcT, class I, class DI>
	__global__ 
	void mult_offset(DstT* dst, const SrcT* weights, const SrcT* orig, DI x, DI y, DI z, I w, I h, I d, I spitch){
		const int xstart = threadIdx.x +blockIdx.x*blockDim.x;
		const int ystart = threadIdx.y +blockIdx.y*blockDim.y;
		const int zstart = threadIdx.z +blockIdx.z*blockDim.z;
		const int xoff   = blockDim.x*gridDim.x;
		const int yoff   = blockDim.y*gridDim.y;
		const int zoff   = blockDim.z*gridDim.z;
		for(I i=zstart;i<d;i+=zoff)
		for(I j=ystart;j<h;j+=yoff)
		for(I k=xstart;k<w;k+=xoff){
			SrcT wgt = weights2d ? weights[k+j*w] : weights[k+j*w+i*w*h];
			dst[k+j*w+i*w*h] += wgt *
				/**PITCH(orig,spitch,clamp(j+y,h)+clamp(i+z,d)*h, clamp(k+x,w) );*/
				/*orig[clamp(j+y,h)*w + clamp(i+z,d)*w*h + clamp(k+x,w)];*/
				tex3D(texref<3,SrcT>::get(), k+x, j+y, i+z);
		}
	}
	template<class DstT, class SrcT, class I, class DI>
	__global__ 
	void get_sqdiff2d(DstT* diffs, const SrcT* src, DI x, DI y, DI z, I w, I h, I d, I spitch){
		const int xstart = threadIdx.x +blockIdx.x*blockDim.x;
		const int ystart = threadIdx.y +blockIdx.y*blockDim.y;
		const int xoff   = blockDim.x*gridDim.x;
		const int yoff   = blockDim.y*gridDim.y;
		for(int j=ystart;j<h;j+=yoff)
			for(int k=xstart;k<w;k+=xoff){
				DstT res = 0.0f;
				for(int i=0;i<d;i+=1){
					/*
					 *DstT v = src[j*w+i*h*w+k]
					 *        -src[clamp(j+y,h)*w+clamp(i+z,d)*w*h+ clamp(k+x,w)];
					 */
					/*
					 *DstT v = *PITCH(src,spitch,j+i*h,k)
					 *        -*PITCH(src,spitch,clamp(j+y,h)+clamp(i+z,d)*h, clamp(k+x,w) );
					 */
					DstT v =  tex3D(texref<3,SrcT>::get(), k, j, i)
						- tex3D(texref<3,SrcT>::get(), k+x,j+y,i+z);
					;
					v = ((k+x>=w) | (j+y>=h)) ? 1E6 : v;
					res += v*v;
				}
				diffs[k+j*w] = res/d;
			}
	}
	template<class DstT, class SrcT, class I, class DI>
	__global__ 
	void get_sqdiff(DstT* diffs, const SrcT* src, DI x, DI y, DI z, I w, I h, I d, I spitch){
		const int xstart = threadIdx.x +blockIdx.x*blockDim.x;
		const int ystart = threadIdx.y +blockIdx.y*blockDim.y;
		const int zstart = threadIdx.z +blockIdx.z*blockDim.z;
		const int xoff   = blockDim.x*gridDim.x;
		const int yoff   = blockDim.y*gridDim.y;
		const int zoff   = blockDim.z*gridDim.z;
		for(int i=zstart;i<d;i+=zoff)
		for(int j=ystart;j<h;j+=yoff)
		for(int k=xstart;k<w;k+=xoff){
			 /*
			  *DstT v = src[j*w+i*h*w+k]
			  *        -src[clamp(j+y,h)*w+clamp(i+z,d)*w*h+ clamp(k+x,w)];
			  */
			/*
			 *DstT v = *PITCH(src,spitch,j+i*h,k)
			 *        -*PITCH(src,spitch,clamp(j+y,h)+clamp(i+z,d)*h, clamp(k+x,w) );
			 */
			DstT v =  tex3D(texref<3,SrcT>::get(), k, j, i)
			       - tex3D(texref<3,SrcT>::get(), k+x,j+y,i+z);
				;
			v = ((k+x>=w) | (j+y>=h) | (i+z>=d)) ? 1E6 : v;
			diffs[k+j*w+i*w*h] = v*v;
		}
	}
	int divup(int a, int b)
	{
		if (a % b)  /* does a divide b leaving a remainder? */
			return a / b + 1; /* add in additional block */
		else
			return a / b; /* divides cleanly */
	}
	
	template<class T>
	void filter_nlmean(cuv::tensor<T,dev_memory_space,row_major>& dst, const cuv::tensor<T,dev_memory_space,row_major>& constsrc, int search_radius, int filter_radius, float sigma, float dist_sigma, float step_size, bool threeDim, bool verbose){
		cuvAssert(!threeDim || constsrc.ndim()==3);

		bool d3 = constsrc.ndim()==3;
		unsigned int w = constsrc.shape()[d3?2:1], h=constsrc.shape()[d3?1:0], d=d3?constsrc.shape()[0]:1;
		const tensor_view<float,dev_memory_space,row_major> src(indices[index_range(0,d)][index_range(0,h)][index_range(0,w)], constsrc);

		if(!equal_shape(dst,src)){
			dst = cuv::tensor<T,dev_memory_space>(src.shape());
		}
		if(0){
			dst = 0.f;
			cuv::tensor<float,host_memory_space> kernel(2*filter_radius+1);
			for(int i=-filter_radius; i<=filter_radius;i++)
				kernel(i+filter_radius) = (float) exp(-i*i);
			kernel /= (float)cuv::sum(kernel);
			kernel = 1.f/kernel.size();
			/*setConvolutionKernel_horizontal(kernel);*/
			/*setConvolutionKernel_vertical(kernel);*/
			/*setConvolutionKernel_depth(kernel);*/
			cuv::tensor<float,dev_memory_space> tmp1(constsrc.shape());
			cuv::tensor<float,dev_memory_space> cpy(constsrc);
			/*convolutionRows   (dst,cpy,filter_radius);*/
			convolutionColumns(dst,cpy,kernel);
			/*convolutionDepth  (dst,cpy,filter_radius);*/
			return;
		}
		cuda_array<T,dev_memory_space> ca(src.shape()[1],src.shape()[2],src.shape()[0],1);
		ca.assign(src);

		typedef typename texref<3,T>::type textype;
		textype tex = texref<3,T>::get();
		hipChannelFormatDesc channelDesc = hipCreateChannelDesc<T>();
		tex.normalized = false;
		tex.filterMode = hipFilterModePoint;
		tex.addressMode[0] = hipAddressModeClamp;
		tex.addressMode[1] = hipAddressModeClamp;
		tex.addressMode[2] = hipAddressModeClamp;
		cuvSafeCall(hipBindTextureToArray(tex, ca.ptr(), channelDesc));

		cuv::tensor<float,dev_memory_space> weights(src.shape());
		cuv::tensor<float,dev_memory_space> diffs(src.shape());
		cuv::tensor<float,dev_memory_space> tmp1(src.shape());

		// prepare kernel
		cuv::tensor<float,host_memory_space> kernel(2*filter_radius+1);
		if(filter_radius!=0){
			kernel = 1.f/kernel.size();
			/*setConvolutionKernel_horizontal(kernel);*/
			/*setConvolutionKernel_vertical(kernel);*/
			/*setConvolutionKernel_depth(kernel);*/
		}

		dst     = (T)0.f;
		weights = (T)0.f;
		typedef float step_type;
		if(threeDim){
			dim3 blocks(divup(w,8),divup(h,8),divup(d,8));
			dim3 threads(8,8,8);
			int fw=(2*search_radius+1)*1.f/step_size;
			ProgressBar pb(fw*fw*fw);
			for(step_type i=-search_radius;i<=search_radius;i+=step_size){
				for(step_type j=-search_radius;j<=search_radius;j+=step_size){
					for(step_type k=-search_radius;k<=search_radius;k+=step_size){
						get_sqdiff<<<blocks,threads>>>(diffs.ptr(),src.ptr(),k,j,i,w,h,d,(unsigned int)src.stride(1));
						if(filter_radius==0){
							tmp1 = diffs;
						}else{
							convolutionRows   (tmp1,diffs,kernel);
							convolutionColumns(diffs,tmp1,kernel);
							convolutionDepth  (tmp1,diffs,kernel);
						}
						tmp1 /= -sigma*sigma;
						if(dist_sigma>0.f)
							tmp1 += -(i*i+j*j+k*k)/(dist_sigma*dist_sigma);
						cuv::apply_scalar_functor(tmp1, SF_EXP);
						weights += tmp1;
						mult_offset<false><<<blocks,threads>>>(dst.ptr(),tmp1.ptr(),src.ptr(),k,j,i,w,h,d,(unsigned int)src.stride(1));
						if(verbose)
							pb.inc();
					}
				}
			}
			if(verbose)
				pb.finish();
			dst /= weights;
		}else{
			dim3 blocks(divup(w,8),divup(h,8),1);
			dim3 threads(16,16,1);
			int fw=(2*search_radius+1)*1.f/step_size;
			ProgressBar pb(fw*fw);
			for(step_type k=-search_radius;k<=search_radius;k+=step_size){
				for(step_type j=-search_radius;j<=search_radius;j+=step_size){
					get_sqdiff2d<<<blocks,threads>>>(diffs.ptr(),src.ptr(),k,j,(step_type)0,w,h,d,(unsigned int)src.stride(0));
					if(filter_radius!=0){
						convolutionRows(tmp1,diffs,kernel);
						convolutionColumns(diffs,tmp1,kernel);
					}
					diffs /= -sigma*sigma;
					if(dist_sigma>0.f)
						diffs += -(j*j+k*k)/(dist_sigma*dist_sigma);
					cuv::apply_scalar_functor(diffs, SF_EXP);
					weights += diffs;
					mult_offset<true><<<blocks,threads>>>(dst.ptr(),diffs.ptr(),src.ptr(),k,j,(step_type)0,w,h,d,(unsigned int)src.stride(0));
					if(verbose)
						pb.inc();
				}
			}
			if(verbose)
				pb.finish();
			tensor_view<float,dev_memory_space> view_w(indices[0][index_range(0,h)][index_range(0,w)],weights);
			for(int l=0;l<d;l++){
				tensor_view<float,dev_memory_space> view_dst(indices[l][index_range(0,h)][index_range(0,w)],dst);
				view_dst/=view_w;
			}
		}
		cuvSafeCall(hipUnbindTexture(tex));
	}

	/*template void filter_nlmean(cuv::tensor<float,dev_memory_space>& dst, const cuv::tensor<float,dev_memory_space>& src);*/
	template void filter_nlmean(cuv::tensor<float,dev_memory_space,row_major>& dst, const cuv::tensor<float,dev_memory_space,row_major>& src, int,int,float,float,float,bool,bool);
}
}
}
