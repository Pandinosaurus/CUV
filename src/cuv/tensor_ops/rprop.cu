#include "hip/hip_runtime.h"
//*LB*
// Copyright (c) 2010, University of Bonn, Institute for Computer Science VI
// All rights reserved.
// 
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
// 
//  * Redistributions of source code must retain the above copyright notice,
//    this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright notice,
//    this list of conditions and the following disclaimer in the documentation
//    and/or other materials provided with the distribution.
//  * Neither the name of the University of Bonn 
//    nor the names of its contributors may be used to endorse or promote
//    products derived from this software without specific prior written
//    permission.
// 
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
// ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
// WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
// FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
// DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
// SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
// OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//*LE*




#include <cuv/tensor_ops/rprop.hpp>

//#define sgn(a) (copysign(1.f,a))
#define sgn(a) ((a==(typeof(a))0) ? 0.f : copysign(1.f,a))

#define DELTA_MAX 5.0f
#define DELTA_MIN (1.0E-8)

#ifdef __CDT_PARSER__
#define __global__
#endif

template<class T, class S>
__global__ void rprop_kernel(T*W, T* dW, S* dW_old, T* rate, int n, T decay, T sparsedecay, T eta_p, T eta_m) {
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int off = blockDim.x * gridDim.x;
	for (unsigned int i = idx; i < n; i += off){
                /*
                        for l1-norm, use ``Orthant-Wise Limited-memory Quasi-Newton Optimizer for L1-regularized Objectives''

			http://research.microsoft.com/en-us/downloads/b1eb1016-1738-4bd5-83a9-370c9d498a03/
                */

		T pg   = -dW[i]; // projected gradient
		T oldW = W[i];
		S sdW  = sgn(pg);
		pg    -= decay * oldW;

		S snW  = sgn(oldW);
		S tmp  = (snW==0) ? sgn(pg) : 0;
		pg    -= snW * sparsedecay;                  // if snW==0, apply to gradient instead...
		pg    -= tmp * min(sparsedecay, fabs(   pg));// ... keeping W at zero!

		S sn = (S)sgn(pg);
		S s  = dW_old[i] * sn;
		T delta=0, step=rate[i];

		if ( s > 0) {
			step = min( eta_p * step, DELTA_MAX);
			delta = sdW * step;
			if(sparsedecay!=0 && delta*pg<=(T)0) // we changed direction while projecting the gradient, don't execute step!
				delta = (T)0;
		}
		else if ( s < 0) {
			step = max( eta_m * step, DELTA_MIN);
			sdW  = 0;
		}
		else {
			if(sparsedecay==(T)0) // do not make a move when sparse decay is on (pg==0)
				delta = sn * step;
		}
		__syncthreads();
		rate[i]   = step;
		dW_old[i] = sdW;
		T newW    = oldW+delta;
		if (sparsedecay == 0.0f)
			W[i] = newW;
		else
			W[i] = (newW*oldW<(T)0) ? (T)0 : newW;
	}
} 

template<class T, class S>
__global__ void rrmsprop_kernel(T*W, T* dW, S* dW_old, T* rate, T* sW, int n, T avg_grad, T delta,  T decay, T sparsedecay, T eta_p, T eta_m, T delta_max, T delta_min) {
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int off = blockDim.x * gridDim.x;
	for (unsigned int i = idx; i < n; i += off){
                /*
                        for l1-norm, use ``Orthant-Wise Limited-memory Quasi-Newton Optimizer for L1-regularized Objectives''

			http://research.microsoft.com/en-us/downloads/b1eb1016-1738-4bd5-83a9-370c9d498a03/
                */

		T pg   = -dW[i]; // projected gradient
		T oldW = W[i];
		S sdW  = sgn(pg);
		pg    -= decay * oldW;

		S snW  = sgn(oldW);
		S tmp  = (snW==0) ? sgn(pg) : 0;
		pg    -= snW * sparsedecay;                  // if snW==0, apply to gradient instead...
		pg    -= tmp * min(sparsedecay, fabs(   pg));// ... keeping W at zero!

		S sn = (S)sgn(pg);
		S s  = dW_old[i] * sn;
		T d=0, step=rate[i];

		if ( s > 0) {
			step = min( eta_p * step, delta_max);
			d = sdW * step;
			if(sparsedecay!=0 && d*pg<=(T)0) // we changed direction while projecting the gradient, don't execute step!
				d = (T)0;
		}
		else if ( s < 0) {
			step = max( eta_m * step, delta_min);
			sdW  = 0;
		}
		else {
			if(sparsedecay==(T)0) // do not make a move when sparse decay is on (pg==0)
				d = sn * step;
		}
		__syncthreads();
		rate[i]   = step;
		dW_old[i] = sdW;
		sW[i]     = avg_grad * sW[i] + (1.f-avg_grad) * dW[i] * dW[i]; //pg*pg;
		T upd     = d / (sqrt(sW[i])+delta);
		T newW    = oldW+upd;
		if (sparsedecay == 0.0f)
			W[i] = newW;
		else
			W[i] = (newW*oldW<(T)0) ? (T)0 : newW;
	}
} 

template<class T>
__global__ void learn_step_weight_decay_kernel(T* A, const T* dA, T alpha, T beta, T sparsedecay, int n) {
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int off = blockDim.x * gridDim.x;
	for (unsigned int i = idx; i < n; i += off){
		A[i] = alpha*dA[i] + beta*A[i];
		/*T f  = alpha*dA[i] + beta*A[i];*/
		/*A[i] = f - sgn(f)*min(sparsedecay,fabs(f));*/
	}
}

template<class T>
__global__ void learn_step_weight_decay_momentum_kernel(T* A, T* M, const T* dA, T lr, T momentum_weight, T l2decay, T sparsedecay, int n) {
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int off = blockDim.x * gridDim.x;
	for (unsigned int i = idx; i < n; i += off){
        T m = M[i];
        m = momentum_weight * m - lr*(dA[i] + l2decay*A[i]);
        A[i] += m;
        M[i] = m;
		/*T f  = alpha*dA[i] + beta*A[i];*/
		/*A[i] = f - sgn(f)*min(sparsedecay,fabs(f));*/
	}
}



namespace cuv{

	template<class V, class S>
	void
	rprop_impl(tensor<V,dev_memory_space>& W, tensor<V,dev_memory_space>& dW, tensor<S,dev_memory_space>& dW_old, tensor<V,dev_memory_space>& rate, V decay, V sparsedecay, V eta_p, V eta_m){
		cuvAssert(decay >= 0);
		cuvAssert(sparsedecay >= 0);
		int num_threads = 512;
		int num_blocks  = min(512,(int)ceil((float)dW.size() / num_threads));
		rprop_kernel<<< num_blocks, num_threads>>>(W.ptr(), dW.ptr(), dW_old.ptr(), rate.ptr(), dW.size(), decay, sparsedecay, eta_p, eta_m);
		cuvSafeCall(hipDeviceSynchronize());
	}

	template<class T, class S>
	void
	rprop_impl(tensor<T,host_memory_space>& W, tensor<T,host_memory_space>& dW, tensor<S,host_memory_space>& dW_old, tensor<T,host_memory_space>& rate, T decay, T sparsedecay, T eta_p, T eta_m){
		cuvAssert(decay >=0);
		cuvAssert(sparsedecay >=0);
		for (unsigned int i = 0; i < dW.size(); i++){
			/*
			   for l1-norm, use ``Orthant-Wise Limited-memory Quasi-Newton Optimizer for L1-regularized Objectives''

				http://research.microsoft.com/en-us/downloads/b1eb1016-1738-4bd5-83a9-370c9d498a03/
			 */

			T pg   = -dW[i]; // projected gradient
			T oldW = W[i];
			S sdW  = sgn(pg);
			pg    -= decay * oldW;

			S snW  = sgn(oldW);
			S tmp  = (snW==0) ? sgn(pg) : 0;
			pg    -= snW * sparsedecay;                  // if snW==0, apply to gradient instead...
			pg    -= tmp * min(sparsedecay, fabs(   pg));// ... keeping W at zero!

			S sn = (S)sgn(pg);
			S s  = dW_old[i] * sn;
			T delta=0, step=rate[i];

			if ( s > 0) {
				step = min( eta_p * step, DELTA_MAX);
				delta = sdW * step;
				if(sparsedecay!=0 && delta*pg<=(T)0) // we changed direction while projecting the gradient, don't execute step!
					delta = (T)0;
			}
			else if ( s < 0) {
				step = max( eta_m * step, DELTA_MIN);
				sdW  = 0;
			}
			else {
				if(sparsedecay==(T)0) // do not make a move when sparse decay is on (pg==0)
					delta = sn * step;
			}
			rate[i]   = step;
			dW_old[i] = sdW;
			T newW    = oldW+delta;
			if (sparsedecay == 0.0f)
				W[i] = newW;
			else
				W[i] = (newW*oldW<(T)0) ? (T)0 : newW;
		}
	}

        template<class __value_type, class __memory_space_type, class S>
	void rprop(tensor<__value_type,__memory_space_type>& W, tensor<__value_type,__memory_space_type>& dW, tensor<S,__memory_space_type>& dW_old, tensor<__value_type,__memory_space_type>& rate, const float& decay, const float& sparsedecay, const float& eta_p, const float& eta_m){
		cuvAssert(dW.ptr());
		cuvAssert(dW_old.ptr());
		cuvAssert(rate.ptr());
		cuvAssert(dW.size() == dW_old.size());
		cuvAssert(dW.size() ==  rate.size());
		rprop_impl(W,dW,dW_old,rate,decay,sparsedecay, eta_p, eta_m);
	}

	template<class V, class S>
	void
	rrmsprop_impl(tensor<V,dev_memory_space>& W, tensor<V,dev_memory_space>& dW, tensor<S,dev_memory_space>& dW_old, tensor<V,dev_memory_space>& rate, tensor<V,dev_memory_space>& sW, V avg_grad, V delta, V decay, V sparsedecay, V eta_p, V eta_m, V delta_max, V delta_min){
		cuvAssert(decay >= 0);
		cuvAssert(sparsedecay >= 0);
		int num_threads = 512;
		int num_blocks  = min(512,(int)ceil((float)dW.size() / num_threads));
		rrmsprop_kernel<<< num_blocks, num_threads>>>(W.ptr(), dW.ptr(), dW_old.ptr(), rate.ptr(), sW.ptr(), dW.size(), avg_grad, delta, decay, sparsedecay, eta_p, eta_m, delta_max, delta_min);
// 		rprop_kernel<<< num_blocks, num_threads>>>(W.ptr(), dW.ptr(), dW_old.ptr(), rate.ptr(), dW.size(), decay, sparsedecay, eta_p, eta_m);
		cuvSafeCall(hipDeviceSynchronize());
	}

	template<class T, class S>
	void
	rrmsprop_impl(tensor<T,host_memory_space>& W, tensor<T,host_memory_space>& dW, tensor<S,host_memory_space>& dW_old, tensor<T,host_memory_space>& rate, tensor<T,host_memory_space>& sW, T avg_grad, T delta, T decay, T sparsedecay, T eta_p, T eta_m, T delta_max, T delta_min){
		cuvAssert(decay >=0);
		cuvAssert(sparsedecay >=0);
		for (unsigned int i = 0; i < dW.size(); i++){
			/*
			   for l1-norm, use ``Orthant-Wise Limited-memory Quasi-Newton Optimizer for L1-regularized Objectives''

				http://research.microsoft.com/en-us/downloads/b1eb1016-1738-4bd5-83a9-370c9d498a03/
			 */

			T pg   = -dW[i]; // projected gradient
			T oldW = W[i];
			S sdW  = sgn(pg);
			pg    -= decay * oldW;

			S snW  = sgn(oldW);
			S tmp  = (snW==0) ? sgn(pg) : 0;
			pg    -= snW * sparsedecay;                  // if snW==0, apply to gradient instead...
			pg    -= tmp * min(sparsedecay, fabs(   pg));// ... keeping W at zero!

			S sn = (S)sgn(pg);
			S s  = dW_old[i] * sn;
			T d=0, step=rate[i];

			if ( s > 0) {
				step = min( eta_p * step, delta_max);
				d = sdW * step;
				if(sparsedecay!=0 && d*pg<=(T)0) // we changed direction while projecting the gradient, don't execute step!
					d = (T)0;
			}
			else if ( s < 0) {
				step = max( eta_m * step, delta_min);
				sdW  = 0;
			}
			else {
				if(sparsedecay==(T)0) // do not make a move when sparse decay is on (pg==0)
					d = sn * step;
			}
			rate[i]   = step;
			dW_old[i] = sdW;
			sW[i]     = avg_grad * sW[i] + (1.f-avg_grad) * dW[i] * dW[i]; // pg*pg;
			T upd     = d / (sqrt(sW[i])+delta);
			T newW    = oldW+upd;
			if (sparsedecay == 0.0f)
				W[i] = newW;
			else
				W[i] = (newW*oldW<(T)0) ? (T)0 : newW;
		}
	}

	template<class __value_type, class __memory_space_type, class S>
	void rrmsprop(tensor<__value_type,__memory_space_type>& W, tensor<__value_type,__memory_space_type>& dW, tensor<S,__memory_space_type>& dW_old, tensor<__value_type,__memory_space_type>& rate, tensor<__value_type,__memory_space_type>& sW, const float& avg_grad, const float& delta, const float& decay, const float& sparsedecay, const float& eta_p, const float& eta_m, const float& delta_max, const float& delta_min){
		cuvAssert(dW.ptr());
		cuvAssert(dW_old.ptr());
		cuvAssert(rate.ptr());
		cuvAssert(sW.ptr());
		cuvAssert(dW.size() == dW_old.size());
		cuvAssert(dW.size() == rate.size());
		cuvAssert(dW.size() == sW.size());
		rrmsprop_impl(W,dW,dW_old,rate,sW,avg_grad,delta,decay,sparsedecay,eta_p,eta_m,delta_max,delta_min);
	}
	
	template<class V>
	void learn_step_weight_decay_impl(tensor<V,dev_memory_space>& W, const tensor<V,dev_memory_space>& dW, const float& alpha, const float& beta, const float& sparsedecay){
		int num_threads = 512;
		int num_blocks  = min(512,(int)ceil((float)dW.size() / num_threads));
		learn_step_weight_decay_kernel<<< num_blocks, num_threads>>>(W.ptr(), dW.ptr(), alpha, beta, sparsedecay, W.size());
		cuvSafeCall(hipDeviceSynchronize());
	}
	template<class V>
	void learn_step_weight_decay_impl(tensor<V,host_memory_space>& W, const tensor<V,host_memory_space>& dW, const float& alpha, const float& beta, const float& sparsedecay){
		const V* dwptr = dW.ptr();
		V* wptr  = W.ptr();
        const unsigned int size = W.size();
		for (unsigned int i = 0; i < size; i++){
			wptr[i]  = alpha*dwptr[i] + beta*wptr[i];
			/*wptr[i] -= sgn(wptr[i])* min(sparsedecay,fabs(wptr[i]));*/
		}
	}
        template<class __value_type, class __memory_space_type>
	void learn_step_weight_decay(tensor<__value_type,__memory_space_type>& W, const tensor<__value_type,__memory_space_type>& dW, const float& learnrate, const float& decay, const float& sparsedecay){
		cuvAssert(dW.ptr());
		cuvAssert(W.ptr());
		cuvAssert(W.size() == dW.size());
		learn_step_weight_decay_impl(W,dW,-learnrate,1.f-learnrate*decay,sparsedecay);
	}

	template<class V>
	void learn_step_weight_decay_momentum_impl(tensor<V,dev_memory_space>& W, tensor<V,dev_memory_space>& momentum, const tensor<V,dev_memory_space>& dW, const float& lr, const float& momentum_weight, const float& l2decay, const float& sparsedecay){
		int num_threads = 512;
		int num_blocks  = min(512,(int)ceil((float)dW.size() / num_threads));
		learn_step_weight_decay_momentum_kernel<<< num_blocks, num_threads >>>(W.ptr(), momentum.ptr(), dW.ptr(), lr, momentum_weight, l2decay, sparsedecay, W.size());
		cuvSafeCall(hipDeviceSynchronize());
	}
	template<class V>
	void learn_step_weight_decay_momentum_impl(tensor<V,host_memory_space>& W, tensor<V,host_memory_space>& momentum,const tensor<V,host_memory_space>& dW, const float& lr, const float& momentum_weight, const float& l2decay, const float& sparsedecay){
		const V* dwptr = dW.ptr();
		V* wptr  = W.ptr();
		V* mptr  = momentum.ptr();
        const unsigned int size = W.size();
		for (unsigned int i = 0; i < size; i++){
            float m = mptr[i];
			m  = momentum_weight * m - lr*(dwptr[i] - l2decay*wptr[i]);
            wptr[i] += m;
            mptr[i] = m;
			/*wptr[i] -= sgn(wptr[i])* min(sparsedecay,fabs(wptr[i]));*/
		}
	}
    template<class V, class M>
	void learn_step_weight_decay_momentum(tensor<V,M>& W, tensor<V,M>& momentum, const tensor<V,M>& dW, const float& learnrate, const float& momentum_weight, const float& decay, const float& sparsedecay){
		cuvAssert(dW.ptr());
		cuvAssert(W.ptr());
		cuvAssert(W.size() == dW.size());
		cuvAssert(W.size() == momentum.size());
		learn_step_weight_decay_momentum_impl(W,momentum,dW,learnrate,momentum_weight,decay,sparsedecay);
	}

#define RPROP_INSTANTIATE(V,S) \
	template void rprop<V,host_memory_space,S>( tensor<V,host_memory_space>&, tensor<V,host_memory_space>&, tensor<S,host_memory_space>&, tensor<V,host_memory_space>&m, const float&, const float&, const float&, const float&); \
	template void rprop<V,dev_memory_space,S>( tensor<V,dev_memory_space>&,  tensor<V,dev_memory_space>&, tensor<S,dev_memory_space>&, tensor<V,dev_memory_space>&, const float&, const float&, const float&, const float&); \
   	template void rrmsprop<V,host_memory_space,S>( tensor<V,host_memory_space>&, tensor<V,host_memory_space>&, tensor<S,host_memory_space>&, tensor<V,host_memory_space>&, tensor<V,host_memory_space>&, const float&, const float&, const float&, const float&, const float&, const float&, const float&, const float&); \
   	template void rrmsprop<V,dev_memory_space,S>( tensor<V,dev_memory_space>&, tensor<V,dev_memory_space>&, tensor<S,dev_memory_space>&, tensor<V,dev_memory_space>&, tensor<V,dev_memory_space>&, const float&, const float&, const float&, const float&, const float&, const float&, const float&, const float&);
#define LSWD_INSTANTIATE(V) \
	template void learn_step_weight_decay( tensor<V,host_memory_space>&, const tensor<V,host_memory_space>&, const float&,const float&, const float&); \
	template void learn_step_weight_decay( tensor<V,dev_memory_space>&,  const tensor<V,dev_memory_space>&, const float&,const float&, const float&); \
	template void learn_step_weight_decay_momentum( tensor<V,host_memory_space>&, tensor<V,host_memory_space>&,const tensor<V,host_memory_space>&, const float&,const float&,const float&, const float&); \
	template void learn_step_weight_decay_momentum( tensor<V,dev_memory_space>&, tensor<V,dev_memory_space>&, const tensor<V,dev_memory_space>&, const float&,const float&,const float&, const float&);

	RPROP_INSTANTIATE(float,float);
	RPROP_INSTANTIATE(float,signed char);
	LSWD_INSTANTIATE(float);

}
