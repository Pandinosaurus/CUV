#include "hip/hip_runtime.h"
//*LB*
// Copyright (c) 2010, University of Bonn, Institute for Computer Science VI
// All rights reserved.
// 
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
// 
//  * Redistributions of source code must retain the above copyright notice,
//    this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright notice,
//    this list of conditions and the following disclaimer in the documentation
//    and/or other materials provided with the distribution.
//  * Neither the name of the University of Bonn 
//    nor the names of its contributors may be used to endorse or promote
//    products derived from this software without specific prior written
//    permission.
// 
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
// ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
// WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
// FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
// DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
// SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
// OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//*LE*


#include <stdexcept>
#include <hipblas.h>
#include <cblas.h>
#include <stdio.h>
#include <float.h>
#include <limits>

#include <thrust/functional.h>

#include <cuv/tools/cuv_general.hpp>
#include <3rd_party/CudaConv/nvmatrix.cuh>
#include <cuv/matrix_ops/matrix_ops.hpp>
#include <cuv/tensor_ops/functors.hpp>

#ifdef __CDT_PARSER__
#define __global__
#define __shared__
#endif

#define PITCH(PTR,PITCH,Y,X) ((typeof(PTR))((PTR) + (PITCH)*(Y)) + (X))
#define CVT_TRANSPOSE(c) \
	(CBLAS_TRANSPOSE)(((c) == 'N' || (c) == 'n') ? CblasNoTrans : \
	 ((c) == 'T' || (c) == 't') ? CblasTrans : \
	 ((c) == 'C' || (c) == 'c') ? CblasConjTrans : \
	 -1)
	/* (mg)the idea is to place the blocks under each other starting at the upper left in the matrix. their threads
	 * add up multiples of their x position (indicated by ty - see above) in shared memory. then we have a 2-dim
	 * array in shared memory that corresponds in size to the block.
	 *
	 * in each block we now have to add up the elements of each row in the shared memory to get the final value. this is done
	 * by logarithmically adding up the elements:
	 * in the first step the second half (in x direction) of the blocks add their values to the first halfs memory locations in
	 * shared memory - then this is repeated for the first half of the threads. a short example for blockDim.y=8 and blockDim.x=1:
	 *
	 * 1st iter(offset=4): a1 a2 a3 a4 | b1 b2 b3 b4
	 *
	 * 2nd iter(offset=2): (a1+b1) (a2+b2) | (a3+b3) (a4+b4)
	 *
	 * 3rd iter(offset=1): ( (a1+b1) + (a3+b3)  ) | ( (a2+b2) +  (a4+b4) )
	 *
	 * 4th iter(offset=0): ( (a1+b1) + (a3+b3)  ) + ( (a2+b2) +  (a4+b4) )
	 *
	 * tx indicates the y-dimension in the matrix; ty indicates the x-dimension in the matrix
	 */

// "coalesced transpose" with no bank conflicts, example from SDK
// potential speedup by 5 possible for "fine-grained transpose"
template<int BLOCK_SIZE, class T, class I>
__global__
void transpose_kernel(T* dst, const T* src, I width, I height,
		I dpitch, I spitch) {
	const I bx = blockIdx.x * blockDim.x;
	const I by = blockIdx.y * blockDim.y;
	I tx = bx + threadIdx.x;
	I ty = by + threadIdx.y;

	__shared__
	T shared[BLOCK_SIZE][BLOCK_SIZE + 1];

	if (tx < width && ty < height) {
		shared[threadIdx.y][threadIdx.x] = *PITCH(src,spitch,ty,tx);
	}
	__syncthreads();

	tx = by + threadIdx.x;
	ty = bx + threadIdx.y;

	if (tx < height && ty < width) {
		*PITCH(dst,dpitch,ty,tx)
		/*dst[(bx + threadIdx.y) * height + by + threadIdx.x]*/
				= shared[threadIdx.x][threadIdx.y];
	}
}
template<int BLOCK_SIZE, class T, class I>
__global__ void transposeNoBankConflicts(T *dst, const T *src,
	       	I width, I height,
		I dpitch, I spitch)
{
	const I BLOCK_ROWS = BLOCK_SIZE;
	__shared__ float tile[BLOCK_SIZE][BLOCK_SIZE+1];

	int xIndex = blockIdx.x * BLOCK_SIZE + threadIdx.x;
	int yIndex = blockIdx.y * BLOCK_SIZE + threadIdx.y;  
	if(xIndex > width) return;

	const T* srcp = PITCH(src,spitch,yIndex,xIndex);

	for (int i=0; i<BLOCK_SIZE; i+=BLOCK_ROWS) {
		tile[threadIdx.y+i][threadIdx.x] = 
			yIndex+i<height ?  *PITCH(srcp,spitch,i,0):0;
	}

	xIndex = blockIdx.y * BLOCK_SIZE + threadIdx.x;
	yIndex = blockIdx.x * BLOCK_SIZE + threadIdx.y;

	T*       dstp = PITCH(dst,dpitch,yIndex,xIndex);
	__syncthreads();

	for (int i=0; i<BLOCK_SIZE; i+=BLOCK_ROWS) {
		if(yIndex+i<width)
			*PITCH(dstp,dpitch,i,0) = tile[threadIdx.x][threadIdx.y+i];
	}
}
namespace cuv {
template<class __value_type, class __memory_space_type, class __index_type>
tensor<__value_type , __memory_space_type,column_major>*blockview(
		tensor<__value_type,__memory_space_type,column_major>& matrix,
				__index_type start_rows,
				__index_type num_rows,
				__index_type start_cols,
				__index_type num_cols,
				column_major
				) {
                        cuvAssert(matrix.ndim()==2);
			cuvAssert(start_rows==0);
			cuvAssert(num_rows==matrix.shape(0))
			return new tensor<__value_type,__memory_space_type,column_major>(indices[index_range(0,num_rows)][index_range(0,num_cols)], matrix.ptr()+matrix.shape(0)*start_cols);
		}

template<class __value_type, class __memory_space_type, class __index_type>
tensor<__value_type,__memory_space_type,row_major>* blockview(
		tensor<__value_type,__memory_space_type,row_major>& matrix,
		__index_type start_rows,
		__index_type num_rows,
		__index_type start_cols,
		__index_type num_cols,
		row_major
) {
        cuvAssert(matrix.ndim()==2);
	cuvAssert(start_cols==0);
	cuvAssert(num_cols==matrix.shape(1))
	return new tensor<__value_type,__memory_space_type,row_major>(indices[index_range(0,num_rows)][index_range(0,num_cols)],matrix.ptr()+matrix.shape(1)*start_rows);
}
template<class __value_type, class __memory_space_type, class __memory_layout, class __index_type>
tensor<__value_type,__memory_space_type,__memory_layout>* blockview(
		tensor<__value_type,__memory_space_type,__memory_layout> & matrix,
		__index_type start_rows,
		__index_type num_rows ,
		__index_type start_cols,
		__index_type num_cols) {
	return blockview(matrix,start_rows,num_rows,start_cols,num_cols, __memory_layout());
}



/// column major blas3
template<>
void prod(tensor<float,dev_memory_space,column_major>& dst,
		const tensor<float,dev_memory_space,column_major>& A,
		const tensor<float,dev_memory_space,column_major>& B,
		char transA,
		char transB,
		const float& factAB,
		const float& factC) {
        cuvAssert(dst.ndim()==2);
        cuvAssert(A.ndim()==2);
        cuvAssert(B.ndim()==2);
	int m = (transA=='t' ? A.shape(1) : A.shape(0));
	int k1 = (transA=='t' ? A.shape(0) : A.shape(1));
	int k2 = (transB=='t' ? B.shape(1) : B.shape(0));
	int n = (transB=='t' ? B.shape(0) : B.shape(1));

	cuvAssert(dst.shape(0) == m);
	cuvAssert(dst.shape(1) == n);
	cuvAssert(k1 == k2);
	cuvAssert(A.ptr());
	cuvAssert(B.ptr());
	cuvAssert(dst.ptr());

	hipblasSgemm(transA, transB, m, n, k1, factAB, A.ptr(), A.shape(0),B.ptr(), B.shape(0), factC, dst.ptr(), dst.shape(0));
	cuvAssert( cublasGetError() == HIPBLAS_STATUS_SUCCESS );
	cuvSafeCall(hipDeviceSynchronize());
}

template<>
void prod(tensor<float,host_memory_space,column_major>& dst,
		const tensor<float,host_memory_space,column_major>& A,
		const tensor<float,host_memory_space,column_major>& B,
		char transA,
		char transB,
		const float& factAB,
		const float& factC) {
        cuvAssert(dst.ndim()==2);
        cuvAssert(A.ndim()==2);
        cuvAssert(B.ndim()==2);

	int m = (transA=='t' ? A.shape(1) : A.shape(0));
	int k1 = (transA=='t' ? A.shape(0) : A.shape(1));
	int k2 = (transB=='t' ? B.shape(1) : B.shape(0));
	int n = (transB=='t' ? B.shape(0) : B.shape(1));

	cuvAssert(dst.shape(0) == m);
	cuvAssert(dst.shape(1) == n);
	cuvAssert(k1 == k2);
	cuvAssert(A.ptr() != NULL);
	cuvAssert(B.ptr() != NULL);
	cuvAssert(dst.ptr());

#if 1 /* CBLAS */
	cblas_sgemm(
			CblasColMajor,
			CVT_TRANSPOSE(transA),
			CVT_TRANSPOSE(transB), m, n, k1,
			factAB, A.ptr(), A.shape(0),B.ptr(), B.shape(0), factC, dst.ptr(), dst.shape(0));
#else /* naive */
	for(int i=0; i<A.shape(0);i++)
	for(int j=0; j<B.shape(1); j++) {
		float f=0;
		for(int k=0;k<A.shape(1);k++) {
			f += A(i,k)*B(k,j);
		}
		dst.set(i,j,f);
	}
#endif
}
/// row major blas3
template<>
void prod(tensor<float,dev_memory_space,row_major>& dst,
		const tensor<float,dev_memory_space,row_major>& A,
		const tensor<float,dev_memory_space,row_major>& B,
		char transA,
		char transB,
		const float& factAB,
		const float& factC) {
        cuvAssert(dst.ndim()==2);
        cuvAssert(A.ndim()==2);
        cuvAssert(B.ndim()==2);
	// we use column major prod and just exchange width and height
	int m = (transB=='t' ? B.shape(0) : B.shape(1));
	int k1 = (transB=='t' ? B.shape(1) : B.shape(0));
	int k2 = (transA=='t' ? A.shape(0) : A.shape(1));
	int n = (transA=='t' ? A.shape(1) : A.shape(0));

	cuvAssert(dst.shape(0) == n);
	cuvAssert(dst.shape(1) == m);
	cuvAssert(k1 == k2);
	cuvAssert(A.ptr());
	cuvAssert(B.ptr());
	cuvAssert(dst.ptr());
	hipblasSgemm(transB, transA, m, n, k1, factAB, B.ptr(), B.shape(1),A.ptr(), A.shape(1), factC, dst.ptr(), dst.shape(1));

	cuvAssert( cublasGetError() == HIPBLAS_STATUS_SUCCESS );
	cuvSafeCall(hipDeviceSynchronize());
}

template<>
void prod(tensor<float,host_memory_space,row_major>& dst,
		const tensor<float,host_memory_space,row_major>& A,
		const tensor<float,host_memory_space,row_major>& B,
		char transA,
		char transB,
		const float& factAB,
		const float& factC) {
        cuvAssert(dst.ndim()==2);
        cuvAssert(A.ndim()==2);
        cuvAssert(B.ndim()==2);
	int m = (transA=='t' ? A.shape(1) : A.shape(0));
	int k1 = (transA=='t' ? A.shape(0) : A.shape(1));
	int k2 = (transB=='t' ? B.shape(1) : B.shape(0));
	int n = (transB=='t' ? B.shape(0) : B.shape(1));

	cuvAssert(dst.shape(0) == m);
	cuvAssert(dst.shape(1) == n);
	cuvAssert(k1 == k2);
	cuvAssert(A.ptr() != NULL);
	cuvAssert(B.ptr() != NULL);
	cuvAssert(dst.ptr());

	cblas_sgemm(
			CblasRowMajor,
			CVT_TRANSPOSE(transA),
			CVT_TRANSPOSE(transB), m, n, k1,
			factAB, A.ptr(), A.shape(1),B.ptr(), B.shape(1), factC, dst.ptr(), dst.shape(1));
}

template<bool UseFactNew, bool UseFactOld, class V, class I, class V2, class OP>
__global__
void matrix_plus_vector_kernel_column_major(V* Dst, const V*Src, const V2* v,I w,I h, OP op, float factNew, float factOld) {
	int tid = blockDim.x*blockIdx.x + threadIdx.x;
	if(tid>h) return;
	V2 tid_v = v[tid];
	for(int i=tid;i<w;i++)
        if(!UseFactOld && !UseFactNew)
            Dst[i] = op(Src[i],tid_v);
        else if(!UseFactOld && UseFactNew)
            Dst[i] = op(Src[i],tid_v)*factNew;
        else if(UseFactOld && !UseFactNew)
            Dst[i] = factOld * Dst[i] + op(Src[i],tid_v);
        else if(UseFactOld && UseFactNew)
            Dst[i] = factOld * Dst[i] + op(Src[i],tid_v)*factNew;
}
template<bool UseFactNew, bool UseFactOld, class V, class I, class V2, class OP>
__global__
void matrix_plus_vector_kernel_column_major2 (V *Dst, const V* Src, const V2* v, I h, I w, OP op, float factNew, float factOld) {
	const unsigned int idx = __mul24(blockIdx.x , blockDim.x) + threadIdx.x;
	const unsigned int numThreads = __mul24(blockDim.x , gridDim.x);

	int stop = w*h;
	for (unsigned int i = idx; i < stop; i += numThreads)
        if(!UseFactNew && !UseFactOld)
            Dst[i] = op(Src[i] , v[i % h]);
        else if(!UseFactNew && UseFactOld)
            Dst[i] = factOld * Dst[i] + op(Src[i] , v[i % h]);
        else if(UseFactNew && !UseFactOld)
            Dst[i] = factNew * op(Src[i] , v[i % h]);
        else if(UseFactNew && UseFactOld)
            Dst[i] = factOld * Dst[i] + factNew * op(Src[i] , v[i % h]);
}
template<bool UseFactNew, bool UseFactOld, class V, class I, class V2, class OP>
__global__
void matrix_plus_vector_kernel_row_major (V *Dst, const V* Src, const V2* v, I h, I w, OP op, float factNew, float factOld) {
	__shared__ V scalar;
	for(unsigned int baseidx = blockIdx.x; baseidx < h; baseidx += gridDim.x) {
		if (threadIdx.x == 0) {
			scalar = (V) v[baseidx];
		}
		__syncthreads();
		for (unsigned int i = threadIdx.x; i < w; i += blockDim.x) {
			const unsigned int k = baseidx * w + i;
            if(!UseFactOld && !UseFactNew)
                Dst[k] = op(Src[k] , scalar);
            else if(!UseFactOld && UseFactNew)
                Dst[k] = op(Src[k] , scalar) * factNew;
            else if(UseFactOld && !UseFactNew)
                Dst[k] = Dst[k] * factOld + op(Src[k] , scalar);
            else if(UseFactOld && UseFactNew)
                Dst[k] = Dst[k] * factOld + op(Src[k] , scalar) * factNew;
		}
		__syncthreads(); // necessary, otherwise the threads use different values of scalar!
	}
}

namespace matrix_op_col_impl {
	template<class V, class V2, class OP>
	void matrix_op_col(tensor<V,dev_memory_space,row_major>& Dst, const tensor<V,dev_memory_space,row_major>& Src, const tensor<V2,dev_memory_space>& v, const OP& op, float factNew, float factOld) {
		cuvAssert(Src.shape(0) == v.size());
        unsigned int other_dim = Src.size()/Src.shape(0);
		const unsigned int num_threads = min(512,other_dim);
		const unsigned int num_blocks  = min(1024,Src.shape(0));
        if(factNew == 1.f && factOld == 0.f)
            matrix_plus_vector_kernel_row_major<false,false><<<num_blocks,num_threads>>>(Dst.ptr(),Src.ptr(), v.ptr(), Src.shape(0), other_dim, op, factNew, factOld);
        else if(factNew == 1.f)
            matrix_plus_vector_kernel_row_major<false,true><<<num_blocks,num_threads>>>(Dst.ptr(),Src.ptr(), v.ptr(), Src.shape(0), other_dim, op, factNew, factOld);
        else if(factOld == 0.f)
            matrix_plus_vector_kernel_row_major<true,false><<<num_blocks,num_threads>>>(Dst.ptr(),Src.ptr(), v.ptr(), Src.shape(0), other_dim, op, factNew, factOld);
        else
            matrix_plus_vector_kernel_row_major<true,true><<<num_blocks,num_threads>>>(Dst.ptr(),Src.ptr(), v.ptr(), Src.shape(0), other_dim, op, factNew, factOld);
		cuvSafeCall(hipDeviceSynchronize());
	}
	template<class V, class V2, class OP>
	void matrix_op_col(tensor<V,dev_memory_space,column_major>& Dst, const tensor<V,dev_memory_space,column_major>& Src, const tensor<V2,dev_memory_space>& v, const OP& op, float factNew, float factOld) {
		cuvAssert(Src.shape(0) == v.size());
        unsigned int other_dim = Src.size()/Src.shape(0);
		const unsigned int num_threads = 512;
		const unsigned int num_blocks  = min(512,(int)ceil((float)Src.size() / num_threads));
        if(factNew == 1.f && factOld == 0.f)
            matrix_plus_vector_kernel_column_major2<false,false><<<num_blocks,num_threads>>>(Dst.ptr(), Src.ptr(), v.ptr(), Src.shape(0), other_dim, op, factNew, factOld);
        else if(                  factOld == 0.f)
            matrix_plus_vector_kernel_column_major2<true,false><<<num_blocks,num_threads>>>(Dst.ptr(), Src.ptr(), v.ptr(), Src.shape(0), other_dim, op, factNew, factOld);
        else if(factNew == 1.f                  )
            matrix_plus_vector_kernel_column_major2<false,true><<<num_blocks,num_threads>>>(Dst.ptr(), Src.ptr(), v.ptr(), Src.shape(0), other_dim, op, factNew, factOld);
        else // if(factNew == 1.f && factOld == 0.f)
            matrix_plus_vector_kernel_column_major2<true,true><<<num_blocks,num_threads>>>(Dst.ptr(), Src.ptr(), v.ptr(), Src.shape(0), other_dim, op, factNew, factOld);
		cuvSafeCall(hipDeviceSynchronize());
	}
	template<class V, class V2, class OP>
	void matrix_op_col(tensor<V,host_memory_space,column_major>& Dst, const tensor<V,host_memory_space,column_major>& Src, const tensor<V2,host_memory_space>& v, const OP& op, float factNew, float factOld) {
		cuvAssert(Src.shape(0) == v.size());
        unsigned int other_dim = Src.size()/Src.shape(0);
		const V2* v_ptr = v.ptr();
		const V * Src_ptr = Src.ptr();
		V * Dst_ptr = Dst.ptr();
        unsigned int Srcshape0 = Src.shape(0);

        // move src ptr, dst ptr and vptr at the same time

        if(factOld == 0.f){
            if(factNew == 1.f)
                for(int j=0;j<other_dim;j++) {
                    v_ptr = v.ptr();
                    for(int i=0;i<Srcshape0;i++,Src_ptr++,v_ptr++,Dst_ptr++)
                        *Dst_ptr = op(*Src_ptr,*v_ptr);
                }
            else
                for(int j=0;j<other_dim;j++) {
                    v_ptr = v.ptr();
                    for(int i=0;i<Srcshape0;i++,Src_ptr++,v_ptr++,Dst_ptr++)
                        *Dst_ptr = factNew * op(*Src_ptr,*v_ptr);
                }
        }else{
            if(factNew == 1.f)
                for(int j=0;j<other_dim;j++) {
                    v_ptr = v.ptr();
                    for(int i=0;i<Srcshape0;i++,Src_ptr++,v_ptr++,Dst_ptr++)
                        *Dst_ptr = *Dst_ptr * factOld + op(*Src_ptr,*v_ptr);
                }
            else
                for(int j=0;j<other_dim;j++) {
                    v_ptr = v.ptr();
                    for(int i=0;i<Srcshape0;i++,Src_ptr++,v_ptr++,Dst_ptr++)
                        *Dst_ptr = *Dst_ptr * factOld + factNew * op(*Src_ptr,*v_ptr);
                }
        }
	}
	template<class V, class V2, class OP>
	void matrix_op_col(tensor<V,host_memory_space,row_major>& Dst, const tensor<V,host_memory_space,row_major>& Src, const tensor<V2,host_memory_space>& v, const OP& op, float factNew, float factOld) {
		cuvAssert(Src.shape(0) == v.size());
        unsigned int other_dim = Src.size()/Src.shape(0);
		const V2* v_ptr = v.ptr();
		const V * Src_ptr = Src.ptr();
		V * Dst_ptr = Dst.ptr();
        unsigned int Srcshape0 = Src.shape(0);

        // in this version, we only move along src+dst horizontally, but keep pos in column vector v constant

        if(factOld == 0.f){
            if(factNew == 1.f )
                for(int i=0;i<Srcshape0;i++, v_ptr++) {
                    for(int j=0;j<other_dim;j++)
                        *Dst_ptr++ = op(*Src_ptr++,*v_ptr);
                }
            else
                for(int i=0;i<Srcshape0;i++, v_ptr++) {
                    for(int j=0;j<other_dim;j++)
                        *Dst_ptr++ = *Dst_ptr * factOld + factNew * op(*Src_ptr++,*v_ptr);
                }
        }else{
            if(factNew == 1.f)
                for(int i=0;i<Srcshape0;i++, v_ptr++) {
                    for(int j=0;j<other_dim;j++)
                        *Dst_ptr++ = *Dst_ptr * factOld + op(*Src_ptr++,*v_ptr);
                }
            else
                for(int i=0;i<Srcshape0;i++, v_ptr++) {
                    for(int j=0;j<other_dim;j++)
                        *Dst_ptr++ = *Dst_ptr * factOld + factNew * op(*Src_ptr++,*v_ptr);
                }
        }
	}
	// ====================  row ======================
	template<class V, class V2, class T, class M, class OP>
	void matrix_op_row(tensor<V,T,M>& Dst, const tensor<V,T,M>& Src, const tensor<V2,T>& v, const OP& op, float factNew, float factOld) {
		cuvAssert(Src.shape(Src.ndim()-1) == v.size());
		matrix_op_col(*(transposed_view(Dst)), *(transposed_view(Src)),v,op, factNew, factOld);
	}

	// ====================  middle ======================
    template<bool UseFactNew, bool UseFactOld,class T, class OP>
       __global__
       void matrix_op_middle_kernel(T* dst, const T* src, const T* v, OP op, const unsigned int dim0, const unsigned int dim1, const unsigned int dim2, float factNew, float factOld){
           unsigned int line = blockIdx.x;
           unsigned int init_offset = line * dim2 * dim1;
           T* dst0;
           const T* src0;
           unsigned int offset_dim0 = gridDim.x * dim2 * dim1;

           for(unsigned int f = 0; f < dim1; f++){
              dst0 = dst + init_offset;
              src0 = src + init_offset;
              line = blockIdx.x;

              T el = v[f];
              unsigned int offset_dim1 = f*dim2;
              for(; line < dim0; line += gridDim.x){
                  /*unsigned int item = threadIdx.x;*/
                  /*for(; item < dim2; item += blockDim.x){*/
                      /*unsigned int index = offset_dim1 + item;*/
                  for(unsigned int index = offset_dim1 + threadIdx.x;
                          index < dim2 + offset_dim1;
                          index += blockDim.x){

                      if(!UseFactOld && !UseFactNew)
                          dst0[index] = op(src0[index], el);
                      else if(!UseFactOld && UseFactNew)
                          dst0[index] = op(src0[index],el)*factNew;
                      else if(UseFactOld && !UseFactNew)
                          dst0[index] = factOld * dst0[index] + op(src0[index],el);
                      else if(UseFactOld && UseFactNew)
                          dst0[index] = factOld * dst0[index] + op(src0[index],el)*factNew;
                  }
                  dst0 += offset_dim0;
                  src0 += offset_dim0;
              }
           }
       }

    template<bool UseFactNew, bool UseFactOld,class T, class OP>
        void matrix_op_middle_host(T* dst, const T* src, const T* v, OP op,  unsigned int dim0, unsigned int dim1, unsigned int dim2, float factNew, float factOld){
            T* dst0;
            const T* src0;
            dst0 = dst;
            src0 = src;
            for(unsigned int f = 0; f < dim1; f++){
                unsigned int offset_dim1 = f * dim2;
                for(unsigned int d0 = 0; d0 < dim0; d0++){
                    unsigned int offset = d0 * dim1 * dim2 + offset_dim1;
                    for(unsigned int d2 = 0; d2 < dim2; ++d2){
                        T el = v[f];
                        unsigned int index = offset + d2;
                        if(!UseFactOld && !UseFactNew)
                            dst0[index] = op(src0[index], el);
                        else if(!UseFactOld && UseFactNew)
                            dst0[index] = op(src0[index],el)*factNew;
                        else if(UseFactOld && !UseFactNew)
                            dst0[index] = factOld * dst0[index] + op(src0[index],el);
                        else if(UseFactOld && UseFactNew)
                            dst0[index] = factOld * dst0[index] + op(src0[index],el)*factNew;
                    }
                }
            }
        }

    template<class V,class M, class T, class OP>
        void matrix_op_middle(tensor<V,M,T>& dst, const tensor<V,M,T>& src, const tensor<V,M,row_major>& v, unsigned int dim, const OP& op, float factNew, float factOld){
            assert(dst.ndim() == src.ndim());
            assert(v.ndim() == 1);
            assert(v.shape(0) == src.shape(dim));

            unsigned int dim0 = 1;
            unsigned int dim1 = src.shape(dim);
            unsigned int dim2 = 1;
            for(unsigned int i = 0; i < src.ndim(); i++){
                if(i < dim)
                    dim0 *= src.shape(i);
                else if(i > dim)
                    dim2 *= src.shape(i);
            }

            if(IsSame<M,host_memory_space>::Result::value){
                if(IsSame<T,row_major>::Result::value){
                    if(factNew == 1.f && factOld == 0.f)
                        matrix_op_middle_host<false,false>(dst.ptr(), src.ptr(), v.ptr(), op, dim0, dim1, dim2, factNew, factOld);
                    else if(                  factOld == 0.f)
                        matrix_op_middle_host<true,false>(dst.ptr(), src.ptr(), v.ptr(), op, dim0, dim1, dim2, factNew, factOld);
                    else if(factNew == 1.f                  )
                        matrix_op_middle_host<false,true>(dst.ptr(), src.ptr(), v.ptr(), op, dim0, dim1, dim2, factNew, factOld);
                    else // if(factNew == 1.f && factOld == 0.f)
                        matrix_op_middle_host<true,true>(dst.ptr(), src.ptr(), v.ptr(), op, dim0, dim1, dim2, factNew, factOld);
                }else{
                    // in the case of column mayor, only dim0 and dim2 are swiched
                    unsigned int temp = dim2;
                    dim2 = dim0;
                    dim0 = temp;

                    if(factNew == 1.f && factOld == 0.f)
                        matrix_op_middle_host<false,false>(dst.ptr(), src.ptr(), v.ptr(), op, dim0, dim1, dim2, factNew, factOld);
                    else if(                  factOld == 0.f)
                        matrix_op_middle_host<true,false>(dst.ptr(), src.ptr(), v.ptr(), op, dim0, dim1, dim2, factNew, factOld);
                    else if(factNew == 1.f                  )
                        matrix_op_middle_host<false,true>(dst.ptr(), src.ptr(), v.ptr(), op, dim0, dim1, dim2, factNew, factOld);
                    else // if(factNew == 1.f && factOld == 0.f)
                        matrix_op_middle_host<true,true>(dst.ptr(), src.ptr(), v.ptr(), op, dim0, dim1, dim2, factNew, factOld);
                }
            }else{
                // device: run kernel
                unsigned int num_threads = min(512, int(32 * ceil(dim2 / 32. )));

                unsigned int num_blocks  = min(1024,dim0);

                if(IsSame<T,row_major>::Result::value){
                    if(factNew == 1.f && factOld == 0.f)
                        matrix_op_middle_kernel<false,false><<<num_blocks,num_threads>>>(dst.ptr(), src.ptr(), v.ptr(), op, dim0, dim1, dim2, factNew, factOld);
                    else if(                  factOld == 0.f)
                        matrix_op_middle_kernel<true,false><<<num_blocks,num_threads>>>(dst.ptr(), src.ptr(), v.ptr(), op, dim0, dim1, dim2, factNew, factOld);
                    else if(factNew == 1.f                  )
                        matrix_op_middle_kernel<false,true><<<num_blocks,num_threads>>>(dst.ptr(), src.ptr(), v.ptr(), op, dim0, dim1, dim2, factNew, factOld);
                    else // if(factNew == 1.f && factOld == 0.f)
                        matrix_op_middle_kernel<true,true><<<num_blocks,num_threads>>>(dst.ptr(), src.ptr(), v.ptr(), op, dim0, dim1, dim2, factNew, factOld);
                }else{
                    // in the case of column mayor, only dim0 and dim2 are swiched
                    unsigned int temp = dim2;
                    dim2 = dim0;
                    dim0 = temp;

                    if(factNew == 1.f && factOld == 0.f)
                        matrix_op_middle_kernel<false,false><<<num_blocks,num_threads>>>(dst.ptr(), src.ptr(), v.ptr(), op, dim0, dim1, dim2, factNew, factOld);
                    else if(                  factOld == 0.f)
                        matrix_op_middle_kernel<true,false><<<num_blocks,num_threads>>>(dst.ptr(), src.ptr(), v.ptr(), op, dim0, dim1, dim2, factNew, factOld);
                    else if(factNew == 1.f                  )
                        matrix_op_middle_kernel<false,true><<<num_blocks,num_threads>>>(dst.ptr(), src.ptr(), v.ptr(), op, dim0, dim1, dim2, factNew, factOld);
                    else // if(factNew == 1.f && factOld == 0.f)
                        matrix_op_middle_kernel<true,true><<<num_blocks,num_threads>>>(dst.ptr(), src.ptr(), v.ptr(), op, dim0, dim1, dim2, factNew, factOld);
                }

                cuvSafeCall(hipDeviceSynchronize());
            }
        }

}

template<class V, class V2, class M, class L>
    void matrix_op_vec(tensor<V,M,L>& Dst, const tensor<V,M,L>& Src, const tensor<V2,M>& v, int axis, BinaryFunctor bf, float factNew, float factOld, int n_params, float param0, float param1){
        if(axis == Src.ndim()-1)
            switch(bf){
                case BF_1ST:
                    matrix_op_col_impl::matrix_op_row(Dst, Src,v, bf_1st<V,V,V2>(), factNew, factOld);
                    break;
                case BF_2ND:
                    matrix_op_col_impl::matrix_op_row(Dst, Src,v, bf_2nd<V,V,V2>(), factNew, factOld);
                    break;
                case BF_ADD:
                    matrix_op_col_impl::matrix_op_row(Dst, Src,v, thrust::plus<V>(), factNew, factOld);
                    break;
                case BF_MULT:
                    matrix_op_col_impl::matrix_op_row(Dst, Src,v, thrust::multiplies<V>(), factNew, factOld);
                    break;
                case BF_DIV:
                    matrix_op_col_impl::matrix_op_row(Dst, Src,v, thrust::divides<V>(), factNew, factOld);
                    break;
                default:
                    throw std::runtime_error("matrix_op_row: binary functor not implemented.");
            }
        else if(axis == 0)
            switch(bf){
                case BF_1ST:
                    matrix_op_col_impl::matrix_op_col(Dst, Src,v, bf_1st<V,V,V2>(), factNew, factOld);
                    break;
                case BF_2ND:
                    matrix_op_col_impl::matrix_op_col(Dst, Src,v, bf_2nd<V,V,V2>(), factNew, factOld);
                    break;
                case BF_ADD:
                    matrix_op_col_impl::matrix_op_col(Dst, Src,v, thrust::plus<V>(), factNew, factOld);
                    break;
                case BF_MULT:
                    matrix_op_col_impl::matrix_op_col(Dst, Src,v, thrust::multiplies<V>(), factNew, factOld);
                    break;
                case BF_DIV:
                    matrix_op_col_impl::matrix_op_col(Dst, Src,v, thrust::divides<V>(), factNew, factOld);
                    break;
                default:
                    throw std::runtime_error("matrix_op_row: binary functor not implemented.");
            }
        else {
            switch(bf){
                case BF_1ST:
                    matrix_op_col_impl::matrix_op_middle(Dst, Src,v, axis,  bf_1st<V,V,V2>(), factNew, factOld);
                    break;
                case BF_2ND:
                    matrix_op_col_impl::matrix_op_middle(Dst, Src,v, axis,  bf_2nd<V,V,V2>(), factNew, factOld);
                    break;
                case BF_ADD:
                    matrix_op_col_impl::matrix_op_middle(Dst, Src,v,axis, thrust::plus<V>(), factNew, factOld);
                    break;
                case BF_MULT:
                    matrix_op_col_impl::matrix_op_middle(Dst, Src,v, axis,thrust::multiplies<V>(), factNew, factOld);
                    break;
                case BF_DIV:
                    matrix_op_col_impl::matrix_op_middle(Dst, Src,v,axis, thrust::divides<V>(), factNew, factOld);
                    break;
                default:
                    throw std::runtime_error("matrix_op_row: binary functor not implemented.");
            }
        }
    }

// ====================  col ======================
template<class __value_type, class __memory_space_type, class __memory_layout_type>
void matrix_plus_col(tensor<__value_type,__memory_space_type,__memory_layout_type>& A, const tensor<__value_type,__memory_space_type>& v) {
   /*matrix_op_col_impl::matrix_op_col(A,A,v, thrust::plus<__value_type>());*/
   matrix_op_vec(A, A, v, 0, BF_ADD);
}
template<class __value_type, class __memory_space_type, class __memory_layout_type>
void matrix_times_col(tensor<__value_type,__memory_space_type,__memory_layout_type>& A, const tensor<__value_type,__memory_space_type>& v) {
	/*matrix_op_col_impl::matrix_op_col(A,A,v, thrust::multiplies<__value_type>());*/
    matrix_op_vec(A, A, v, 0, BF_MULT);
}
template<class __value_type, class __memory_space_type, class __memory_layout_type>
void matrix_divide_col(tensor<__value_type,__memory_space_type,__memory_layout_type>& A, const tensor<__value_type,__memory_space_type>& v) {
	/*matrix_op_col_impl::matrix_op_col(A,A,v, thrust::divides<__value_type>());*/
    matrix_op_vec(A, A, v, 0, BF_DIV);
}
// ====================  row ======================
template<class __value_type, class __memory_space_type, class __memory_layout_type>
void matrix_plus_row(tensor<__value_type,__memory_space_type,__memory_layout_type>& A, const tensor<__value_type,__memory_space_type>& v) {
   /*matrix_op_col_impl::matrix_op_row(A,A,v, thrust::plus<__value_type>());*/
   matrix_op_vec(A, A, v, A.ndim()-1, BF_ADD);
}
template<class __value_type, class __memory_space_type, class __memory_layout_type>
void matrix_times_row(tensor<__value_type,__memory_space_type,__memory_layout_type>& A, const tensor<__value_type,__memory_space_type>& v) {
	/*matrix_op_col_impl::matrix_op_row(A,A,v, thrust::multiplies<__value_type>());*/
    matrix_op_vec(A, A, v, A.ndim()-1, BF_MULT);
}
template<class __value_type, class __memory_space_type, class __memory_layout_type>
void matrix_divide_row(tensor<__value_type,__memory_space_type,__memory_layout_type>& A, const tensor<__value_type,__memory_space_type>& v) {
	/*matrix_op_col_impl::matrix_op_row(A,A,v, thrust::divides<__value_type>());*/
    matrix_op_vec(A, A, v, A.ndim()-1, BF_DIV);
}

namespace transpose_impl{
	template<class V>
	void transpose(tensor<V, dev_memory_space, column_major>& dst,
			 const tensor<V, dev_memory_space, column_major>& src) {
                cuvAssert(dst.ndim()==2);
                cuvAssert(src.ndim()==2);
		cuvAssert(dst.shape(1) == src.shape(0));
		cuvAssert(dst.shape(0) == src.shape(1));
                typedef typename tensor<V, dev_memory_space, column_major>::index_type I;
		const I width = src.shape(0);
		const I height = src.shape(1);
		static const int BLOCK_SIZE = 16;
		const int numBlocksX = ceil((float)width / BLOCK_SIZE);
		const int numBlocksY = ceil((float)height / BLOCK_SIZE);
		dim3 gridSize(numBlocksX, numBlocksY, 1);
		dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE, 1);
		transpose_kernel<BLOCK_SIZE><<<gridSize, blockSize>>>(dst.ptr(), src.ptr(), width, height,dst.stride(1),src.stride(1));
		cuvSafeCall(hipDeviceSynchronize());
	}

	template<class V>
	void transpose(tensor<V,dev_memory_space,row_major>& dst,
			 const tensor<V,dev_memory_space,row_major>& src) {
                cuvAssert(dst.ndim()==2);
                cuvAssert(src.ndim()==2);
		cuvAssert(dst.shape(1) == src.shape(0));
		cuvAssert(dst.shape(0) == src.shape(1));
                typedef typename tensor<V, dev_memory_space, row_major>::index_type I;
		const I width = src.shape(1);
		const I height = src.shape(0);
		static const int BLOCK_SIZE = 16;
		const int numBlocksX = ceil((float)width / BLOCK_SIZE);
		const int numBlocksY = ceil((float)height / BLOCK_SIZE);
		dim3 gridSize(numBlocksX, numBlocksY, 1);
		dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE, 1);
		transpose_kernel<BLOCK_SIZE><<<gridSize, blockSize>>>(dst.ptr(), src.ptr(), width, height,dst.stride(0),src.stride(0));
		cuvSafeCall(hipDeviceSynchronize());
	}

	template<class V>
	void transpose(tensor<V,host_memory_space,column_major>& dst,
			 const tensor<V,host_memory_space,column_major>& src) {
                cuvAssert(dst.ndim()==2);
                cuvAssert(src.ndim()==2);
		cuvAssert(dst.shape(1) == src.shape(0));
		cuvAssert(dst.shape(0) == src.shape(1));
		V* dst_ptr = dst.ptr();
		const V* src_ptr = src.ptr();
		for(int i=0; i<dst.shape(1); i++) {
			for(int j=0; j<dst.shape(0); j++) {
				*dst_ptr++ = src_ptr[j*src.shape(0)];
			}
			src_ptr++;
		}
	}

	template<class V>
	void transpose(tensor<V,host_memory_space,row_major>& dst,
			 const tensor<V,host_memory_space,row_major>& src) {
                cuvAssert(dst.ndim()==2);
                cuvAssert(src.ndim()==2);
		cuvAssert(dst.shape(1) == src.shape(0));
		cuvAssert(dst.shape(0) == src.shape(1));
		V* dst_ptr = dst.ptr();
		const V* src_ptr = src.ptr();
		for(int i=0; i<dst.shape(0); i++) {
			for(int j=0; j<dst.shape(1); j++) {
				*dst_ptr++ = src_ptr[j*src.shape(1)];
			}
			src_ptr++;
		}
	}
} // namespace transpose_impl

template<class __value_type, class __memory_space_type, class __memory_layout_type>
void transpose(tensor<__value_type,__memory_space_type, __memory_layout_type>& dst, const tensor<__value_type,__memory_space_type, __memory_layout_type>& src){
	transpose_impl::transpose(dst,src);
}

template<class V, class T, class M>
cuv::tensor<V,T,typename other_memory_layout<M>::type> * transposed_view_p(cuv::tensor<V,T,M>&  src){
        std::vector<unsigned int> shape = src.shape();
        std::reverse(shape.begin(), shape.end());
	return new tensor<V,T,typename other_memory_layout<M>::type>(shape,src.ptr());
}

template<class V, class T, class M>
const cuv::tensor<V,T,typename other_memory_layout<M>::type> * transposed_view_p(const cuv::tensor<V,T,M>&  src){
        std::vector<unsigned int> shape = src.shape();
        std::reverse(shape.begin(), shape.end());
	return new tensor<V,T,typename other_memory_layout<M>::type>(shape,const_cast<V*>(src.ptr()));
}

#define INSTANTIATE_MOV(V1,V2,M) \
  template void matrix_op_vec(tensor<V1,dev_memory_space,M>&, const tensor<V1,dev_memory_space,M>&, const tensor<V2,dev_memory_space>&, int, BinaryFunctor, float,float, int, float,float);   \
  template void matrix_op_vec(tensor<V1,host_memory_space,M>&, const tensor<V1,host_memory_space,M>&, const tensor<V2,host_memory_space>&, int, BinaryFunctor, float,float, int, float,float);   \

#define INSTANTIATE_MV(V1,V2,M) \
  template void matrix_plus_col(tensor<V1,dev_memory_space,M>&, const tensor<V2,dev_memory_space>&);   \
  template void matrix_plus_col(tensor<V1,host_memory_space,M>&, const tensor<V2,host_memory_space>&); \
  template void matrix_times_col(tensor<V1,dev_memory_space,M>&, const tensor<V2,dev_memory_space>&);  \
  template void matrix_times_col(tensor<V1,host_memory_space,M>&, const tensor<V2,host_memory_space>&); \
  template void matrix_divide_col(tensor<V1,dev_memory_space,M>&, const tensor<V2,dev_memory_space>&);  \
  template void matrix_divide_col(tensor<V1,host_memory_space,M>&, const tensor<V2,host_memory_space>&); \
  template void matrix_plus_row(tensor<V1,dev_memory_space,M>&, const tensor<V2,dev_memory_space>&);   \
  template void matrix_plus_row(tensor<V1,host_memory_space,M>&, const tensor<V2,host_memory_space>&); \
  template void matrix_times_row(tensor<V1,dev_memory_space,M>&, const tensor<V2,dev_memory_space>&);  \
  template void matrix_times_row(tensor<V1,host_memory_space,M>&, const tensor<V2,host_memory_space>&); \
  template void matrix_divide_row(tensor<V1,dev_memory_space,M>&, const tensor<V2,dev_memory_space>&);  \
  template void matrix_divide_row(tensor<V1,host_memory_space,M>&, const tensor<V2,host_memory_space>&);


#define INSTANTIATE_BLOCKVIEW(V,M,I) \
  template tensor<V,host_memory_space,M>* blockview(tensor<V,host_memory_space,M>&,I,I,I,I); \
  template tensor<V,dev_memory_space,M>* blockview(tensor<V,dev_memory_space,M>&,I,I,I,I);

#define INSTANTIATE_TRANSPOSE(V,M) \
  template void transpose(tensor<V, host_memory_space, M>&, const tensor<V, host_memory_space, M>&); \
  template void transpose(tensor<V, dev_memory_space , M>&, const tensor<V, dev_memory_space , M>&);

#define INSTANTIATE_TRANSPOSED_VIEW(V) \
  template tensor<V,host_memory_space,other_memory_layout<column_major>::type >* transposed_view_p(tensor<V,host_memory_space,column_major>&);\
  template tensor<V,host_memory_space,other_memory_layout<row_major>::type >* transposed_view_p(tensor<V,host_memory_space,row_major>&);\
  template tensor<V,dev_memory_space,other_memory_layout<column_major>::type >* transposed_view_p(tensor<V,dev_memory_space,column_major>&);\
  template tensor<V,dev_memory_space,other_memory_layout<row_major>::type >* transposed_view_p(tensor<V,dev_memory_space,row_major>&);\
  template const tensor<V,host_memory_space,other_memory_layout<column_major>::type >* transposed_view_p(const tensor<V,host_memory_space,column_major>&);\
  template const tensor<V,host_memory_space,other_memory_layout<row_major>::type >* transposed_view_p(const tensor<V,host_memory_space,row_major>&);\
  template const tensor<V,dev_memory_space,other_memory_layout<column_major>::type >* transposed_view_p(const tensor<V,dev_memory_space,column_major>&);\
  template const tensor<V,dev_memory_space,other_memory_layout<row_major>::type >* transposed_view_p(const tensor<V,dev_memory_space,row_major>&);

INSTANTIATE_TRANSPOSE(float,column_major);
INSTANTIATE_TRANSPOSE(float,row_major);
INSTANTIATE_TRANSPOSE(int,column_major);
INSTANTIATE_TRANSPOSE(int,row_major);
INSTANTIATE_TRANSPOSE(unsigned char,column_major);
INSTANTIATE_TRANSPOSE(unsigned char,row_major);

INSTANTIATE_TRANSPOSED_VIEW(float);
INSTANTIATE_TRANSPOSED_VIEW(int);
INSTANTIATE_TRANSPOSED_VIEW(unsigned int);
INSTANTIATE_TRANSPOSED_VIEW(char);
INSTANTIATE_TRANSPOSED_VIEW(unsigned char);

INSTANTIATE_MOV(float, float, column_major);
INSTANTIATE_MOV(float, float, row_major);

INSTANTIATE_MV(float, float, column_major);
INSTANTIATE_MV(float, float, row_major);
/*INSTANTIATE_MV(float, unsigned char, column_major);*/
/*INSTANTIATE_MV(float, unsigned char, row_major);*/

INSTANTIATE_BLOCKVIEW(float,column_major,int);
INSTANTIATE_BLOCKVIEW(float,row_major,int);


}; // cuv
